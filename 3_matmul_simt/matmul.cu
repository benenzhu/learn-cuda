
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdio.h>

#define PRINT_IF(cond, ...) if (cond) printf(__VA_ARGS__);

__host__ __device__ constexpr int cdiv(int a, int b) { return (a + b - 1) / b; }

constexpr int WARP_SIZE = 32;

// naive kernel. 1 row dot 1 column
__global__ void matmul_v1_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
  const int col = blockIdx.x * blockDim.x + threadIdx.x;
  const int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (row >= M || col >= N)
    return;

  float total = 0.0f;

  // broadcast read from A since each warp reads the same A value
  // coalesce read from B since each warp reads consecutive B values
  for (int k = 0; k < K; k++)
    total += A[row * K + k] * B[k * N + col];

  // coalesce write to C since each warp writes consecutive C values
  C[row * N + col] = total;
}

void matmul_v1(const float *A, const float *B, float *C, int M, int N, int K) {
  // determine optimal block size at runtime
  int block_size_total;
  int min_grid_size; // we don't need this
  hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size_total, matmul_v1_kernel, 0, 0);

  // NOTE: blockIdx.x is the fastest changing dimension. thus, we assign column index to it
  // intuitively, block dimensions will be PyTorch's dimensions in reverse.
  // NOTE: blockDim.x must be multiple of 32 (warpSize) to ensure coalesce memory access
  dim3 block_size(WARP_SIZE, block_size_total / WARP_SIZE);
  dim3 grid_size(cdiv(N, WARP_SIZE), cdiv(M, block_size.y));
  matmul_v1_kernel<<<grid_size, block_size>>>(A, B, C, M, N, K);
}

// read 2D block into shared memory for caching
template <int BLOCK_SIZE>
__global__ void matmul_v2_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
  const int tid_x = threadIdx.x;
  const int tid_y = threadIdx.y;

  const int offset_m = blockIdx.y * BLOCK_SIZE;
  const int offset_n = blockIdx.x * BLOCK_SIZE;

  A += offset_m * K;             // skip x rows
  B += offset_n;                 // skip y columns
  C += offset_m * N + offset_n;  // skip x rows and y columns

  // we cannot return early since all threads need to synchronize
  __shared__ float A_shmem[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float B_shmem[BLOCK_SIZE][BLOCK_SIZE];
  float acc = 0.0f;

  // we move block by block along K dim
  for (int offset_k = 0; offset_k < K; offset_k += BLOCK_SIZE) {
    // load data from global memory (DDR/HBM) to shared memory (SRAM)
    // notice now each thread only loads 2 x n_blocks elements
    // coalesced memory read for both A and B
    A_shmem[tid_y][tid_x] = tid_y < (M - offset_m) && tid_x < (K - offset_k) ? A[tid_y * K + tid_x] : 0.0f;
    B_shmem[tid_y][tid_x] = tid_y < (K - offset_k) && tid_x < (N - offset_n) ? B[tid_y * N + tid_x] : 0.0f;

    // wait for all threads in a block to load data
    __syncthreads();

    // compute from shared memory
    for (int k = 0; k < BLOCK_SIZE; k++)
      acc += A_shmem[tid_y][k] * B_shmem[k][tid_x];

    // wait to finish before moving to the next tile
    __syncthreads();

    A += BLOCK_SIZE;      // stride 1 in K dim
    B += BLOCK_SIZE * N;  // stride N in K dim
  }

  if (tid_y < (M - offset_m) && tid_x < (N - offset_n))
    C[tid_y * N + tid_x] = acc;
}

void matmul_v2(const float *A, const float *B, float *C, int M, int N, int K) {
  // we can't use a larger block size since we are limited by 1024 threads per block
  constexpr int BLOCK_SIZE = 32;
  dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid_size(cdiv(N, BLOCK_SIZE), cdiv(M, BLOCK_SIZE));
  matmul_v2_kernel<BLOCK_SIZE><<<grid_size, block_size>>>(A, B, C, M, N, K);
}

// we want to load a (HEIGHT, WIDTH) tile from global to shared memory.
// just load a BLOCK_SIZE of data until the whole tile is loaded.
template <int BLOCK_SIZE, int HEIGHT, int WIDTH>
__device__ void load_shmem(const float *in, int in_row_stride, int in_max_row, int in_max_col,
                           float out[HEIGHT][WIDTH], int tid) {
  for (int idx = tid; idx < HEIGHT * WIDTH; idx += BLOCK_SIZE) {
    const int row = idx / WIDTH;
    const int col = idx % WIDTH;
    out[row][col] = row < in_max_row && col < in_max_col ? in[row * in_row_stride + col] : 0.0f;
  }
}

// thread coarsening
template <int BLOCK_SIZE, int BLOCK_M, int BLOCK_N, int BLOCK_K>
__global__ void matmul_v3_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
  const int tid = threadIdx.x;
  const int block_id = blockIdx.x;

  // assign block linearly
  const int grid_width = cdiv(N, BLOCK_N);
  const int block_id_m = block_id / grid_width;
  const int block_id_n = block_id % grid_width;

  const int offset_m = block_id_m * BLOCK_M;
  const int offset_n = block_id_n * BLOCK_N;

  A += offset_m * K;
  B += offset_n;
  C += offset_m * N + offset_n;

  __shared__ float A_shmem[BLOCK_M][BLOCK_K];
  __shared__ float B_shmem[BLOCK_K][BLOCK_N];

  // each thread is responsible for (BLOCK_M * BLOCK_N / BLOCK_SIZE) output elements
  float acc[BLOCK_M * BLOCK_N / BLOCK_SIZE] = {0.0f};

  // we move block by block along K dim
  for (int offset_k = 0; offset_k < K; offset_k += BLOCK_K) {
    // decouple global memory read, so we don't need to care about assigning which thread
    // to read which element.
    // load (BLOCK_M, BLOCK_K) from A and (BLOCK_K, BLOCK_N) from B
    load_shmem<BLOCK_SIZE, BLOCK_M, BLOCK_K>(A, K, M - offset_m, K - offset_k, A_shmem, tid);
    load_shmem<BLOCK_SIZE, BLOCK_K, BLOCK_N>(B, N, K - offset_k, N - offset_n, B_shmem, tid);
    __syncthreads();

    // do a mini matmul of (BLOCK_M, BLOCK_K) x (BLOCK_K, BLOCK_N) = (BLOCK_M, BLOCK_N)
    // simply assign a BLOCK_SIZE of threads to a BLOCK_SIZE of elements in output tile
    for (int idx = tid; idx < BLOCK_M * BLOCK_N; idx += BLOCK_SIZE) {
      const int local_idx = idx / BLOCK_SIZE;
      const int col = idx % BLOCK_N;
      const int row = idx / BLOCK_N;

      for (int k = 0; k < BLOCK_K; k++)
        acc[local_idx] += A_shmem[row][k] * B_shmem[k][col];
    }
    __syncthreads();

    A += BLOCK_K;
    B += BLOCK_K * N;
  }

  // write (BLOCK_M, BLOCK_N) to C
  for (int idx = tid; idx < BLOCK_M * BLOCK_N; idx += BLOCK_SIZE) {
    const int local_idx = idx / BLOCK_SIZE;
    const int row = idx / BLOCK_N;
    const int col = idx % BLOCK_N;

    if (row < (M - offset_m) && col < (N - offset_n))
      C[row * N + col] = acc[local_idx];
  }
}

void matmul_v3(const float *A, const float *B, float *C, int M, int N, int K) {
  // we are limited by the amount of shared memory
  // 128 * 32 * 2 * 4 = 32kB
  const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 32;
  const int BLOCK_SIZE = 256;
  const int grid_size = cdiv(M, BLOCK_M) * cdiv(N, BLOCK_N);
  matmul_v3_kernel<BLOCK_SIZE, BLOCK_M, BLOCK_N, BLOCK_K><<<grid_size, BLOCK_SIZE>>>(A, B, C, M, N, K);
}

// register cache with 2D thread tiling
// only mini matmul is different from v3
template <int BLOCK_SIZE, int BLOCK_M, int BLOCK_N, int BLOCK_K, int THREAD_N, bool VECTORIZED_WRITE>
__global__ void matmul_v4_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
  const int tid = threadIdx.x;
  const int block_id = blockIdx.x;

  const int block_grid_width = cdiv(N, BLOCK_N);
  const int block_id_m = block_id / block_grid_width;
  const int block_id_n = block_id % block_grid_width;

  const int offset_m = block_id_m * BLOCK_M;
  const int offset_n = block_id_n * BLOCK_N;

  A += offset_m * K;
  B += offset_n;

  __shared__ float A_shmem[BLOCK_M][BLOCK_K];
  __shared__ float B_shmem[BLOCK_K][BLOCK_N];

  // each thread will calculate (THREAD_M, THREAD_N) thread-tile of output (BLOCK_M, BLOCK_N) block-tile
  constexpr int thread_tile_size = BLOCK_M * BLOCK_N / BLOCK_SIZE;
  constexpr int THREAD_M = thread_tile_size / THREAD_N;
  float acc[THREAD_M][THREAD_N] = {0.0f};

  const int thread_tile_grid_width = BLOCK_N / THREAD_N;
  const int thread_tile_id_m = tid / thread_tile_grid_width;
  const int thread_tile_id_n = tid % thread_tile_grid_width;

  const int thread_tile_offset_m = thread_tile_id_m * THREAD_M;
  const int thread_tile_offset_n = thread_tile_id_n * THREAD_N;

  const float *A_thread_tile = reinterpret_cast<const float *>(A_shmem) + thread_tile_offset_m * BLOCK_K;
  const float *B_thread_tile = reinterpret_cast<const float *>(B_shmem) + thread_tile_offset_n;

  for (int offset_k = 0; offset_k < K; offset_k += BLOCK_K) {
    load_shmem<BLOCK_SIZE, BLOCK_M, BLOCK_K>(A, K, M - offset_m, K - offset_k, A_shmem, tid);
    load_shmem<BLOCK_SIZE, BLOCK_K, BLOCK_N>(B, N, K - offset_k, N - offset_n, B_shmem, tid);
    __syncthreads();

    // mini-matmul with thread-tile
    // notice that we put k as the outermost loop.
    // column of A_thread_tile and row of B_thread_tile is cached to A_reg[] and B_reg[].
    // there is shared memory bank conflict
    for (int k = 0; k < BLOCK_K; k++) {
      float A_reg[THREAD_M];
      float B_reg[THREAD_N];

      for (int m = 0; m < THREAD_M; m++)
        A_reg[m] = A_thread_tile[m * BLOCK_K + k];
      
      for (int n = 0; n < THREAD_N; n++)
        B_reg[n] = B_thread_tile[k * BLOCK_N + n];

      for (int m = 0; m < THREAD_M; m++)
        for (int n = 0; n < THREAD_N; n++)
          acc[m][n] += A_reg[m] * B[n];
    }
    __syncthreads();

    A += BLOCK_K;
    B += BLOCK_K * N;
  }

  C += (offset_m + thread_tile_offset_m) * N + (offset_n + thread_tile_offset_n);

  // uncoalesced memory write
  // fixing it doesn't seem to make the kernel faster.
  if (!VECTORIZED_WRITE) {
    for (int m = 0; m < THREAD_M; m++)
      for (int n = 0; n < THREAD_N; n++)
        if (m < (M - (offset_m + thread_tile_offset_m)) && n < (N - (offset_n + thread_tile_offset_n)))
          C[m * N + n] = acc[m][n];

  } else {
    // using vectorized write will help with uncoalesced memory write (issue fewer txn).
    float4 *C_float4 = reinterpret_cast<float4 *>(C);

    for (int m = 0; m < THREAD_M; m++) {
      for (int n = 0; n < THREAD_N; n += 4) {
        float4 tmp = {acc[m][n], acc[m][n+1], acc[m][n+2], acc[m][n+3]};

        // TODO: handle n % 4 != 0
        if (m < (M - (offset_m + thread_tile_offset_m)) && n < (N - (offset_n + thread_tile_offset_n)))
          C_float4[(m * N + n) / 4] = tmp;
      }
    }
  }
}

void matmul_v4_1(const float *A, const float *B, float *C, int M, int N, int K) {
  const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 32;
  const int THREAD_N = 32;  // THREAD_M will be 2
  const int BLOCK_SIZE = 256;
  const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
  matmul_v4_kernel<BLOCK_SIZE, BLOCK_M, BLOCK_N, BLOCK_K, THREAD_N, false><<<grid_size, BLOCK_SIZE>>>(A, B, C, M, N, K);
}

void matmul_v4_2(const float *A, const float *B, float *C, int M, int N, int K) {
  const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 32;
  const int THREAD_N = 32;  // THREAD_M will be 2
  const int BLOCK_SIZE = 256;
  const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
  matmul_v4_kernel<BLOCK_SIZE, BLOCK_M, BLOCK_N, BLOCK_K, THREAD_N, true><<<grid_size, BLOCK_SIZE>>>(A, B, C, M, N, K);
}

// warp tiling
// we don't actually use MMA instruction here. but to follow the terminologies used by cutlass
// https://github.com/NVIDIA/cutlass/blob/main/media/docs/efficient_gemm.md
// we name the variables as MMA_M and MMA_N, which is tiling of a warp within a warp tile.
template <int BLOCK_SIZE, int BLOCK_M, int BLOCK_N, int BLOCK_K, int WARP_N, int MMA_M, int MMA_N, int THREAD_N>
__global__ void matmul_v5_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
  const int tid = threadIdx.x;
  const int block_id = blockIdx.x;

  const int block_grid_width = cdiv(N, BLOCK_N);
  const int block_id_m = block_id / block_grid_width;
  const int block_id_n = block_id % block_grid_width;

  const int offset_m = block_id_m * BLOCK_M;
  const int offset_n = block_id_n * BLOCK_N;

  A += offset_m * K;
  B += offset_n;

  __shared__ float A_shmem[BLOCK_M][BLOCK_K];
  __shared__ float B_shmem[BLOCK_K][BLOCK_N];

  // each warp will calculate (WARP_M, WARP_N) tile of output (BLOCK_M, BLOCK_N) tile
  constexpr int num_warps = BLOCK_SIZE / WARP_SIZE;
  constexpr int warp_tile_size = BLOCK_M * BLOCK_N / num_warps;
  constexpr int WARP_M = warp_tile_size / WARP_N;

  constexpr int warp_grid_width = BLOCK_N / WARP_N;
  const int warp_id = tid / WARP_SIZE;
  const int warp_id_m = warp_id / warp_grid_width;
  const int warp_id_n = warp_id % warp_grid_width;

  // each warp will iterate over (WARP_ITER_M, WARP_ITER_N) of (MMA_M, MMA_N) tiles
  static_assert(WARP_M % MMA_M == 0);
  static_assert(WARP_N % MMA_N == 0);
  constexpr int WARP_ITER_M = WARP_M / MMA_M;
  constexpr int WARP_ITER_N = WARP_N / MMA_N;

  // each thread will calculate (THREAD_M, THREAD_N) tile of (MMA_M, MMA_N) tile
  static_assert(MMA_M * MMA_N % WARP_SIZE == 0);
  static_assert((MMA_M * MMA_N / WARP_SIZE) % THREAD_N == 0);
  constexpr int thread_tile_size = MMA_M * MMA_N / WARP_SIZE;
  constexpr int THREAD_M = thread_tile_size / THREAD_N;

  static_assert(MMA_N % THREAD_N == 0);
  constexpr int thread_tile_grid_width = MMA_N / THREAD_N;
  const int lane_id = tid % WARP_SIZE;
  const int lane_id_m = lane_id / thread_tile_grid_width;
  const int lane_id_n = lane_id % thread_tile_grid_width;

  // each thread will calculate (THREAD_M, THREAD_N) tile of (MMA_M, MMA_N) tile
  // there are (WARP_ITER_M, WARP_ITER_N) of (MMA_M, MMA_N) tiles in each warp tile
  float acc[WARP_ITER_M][WARP_ITER_N][THREAD_M][THREAD_N] = {0.0f};

  for (int offset_k = 0; offset_k < K; offset_k += BLOCK_K) {
    load_shmem<BLOCK_SIZE, BLOCK_M, BLOCK_K>(A, K, M - offset_m, K - offset_k, A_shmem, tid);
    load_shmem<BLOCK_SIZE, BLOCK_K, BLOCK_N>(B, N, K - offset_k, N - offset_n, B_shmem, tid);
    __syncthreads();

    for (int k = 0; k < BLOCK_K; k++) {
      float A_reg[WARP_ITER_M][THREAD_M];
      float B_reg[WARP_ITER_N][THREAD_N];

      for (int warp_iter_m = 0; warp_iter_m < WARP_ITER_M; warp_iter_m++)
        for (int m = 0; m < THREAD_M; m++) {
          const int row = warp_id_m * WARP_M + warp_iter_m * MMA_M + lane_id_m * THREAD_M + m;
          A_reg[warp_iter_m][m] = A_shmem[row][k];
        }

      for (int warp_iter_n = 0; warp_iter_n < WARP_ITER_N; warp_iter_n++)
        for (int n = 0; n < THREAD_N; n++) {
          const int col = warp_id_n * WARP_N + warp_iter_n * MMA_N + lane_id_n * THREAD_N + n;
          B_reg[warp_iter_n][n] = B_shmem[k][col];
        }

      for (int warp_iter_m = 0; warp_iter_m < WARP_ITER_M; warp_iter_m++)
        for (int warp_iter_n = 0; warp_iter_n < WARP_ITER_N; warp_iter_n++)
          for (int m = 0; m < THREAD_M; m++)
            for (int n = 0; n < THREAD_N; n++)
              acc[warp_iter_m][warp_iter_n][m][n] += A_reg[warp_iter_m][m] * B_reg[warp_iter_n][n];
    }
    __syncthreads();

    A += BLOCK_K;
    B += BLOCK_K * N;
  }

  C += offset_m * N + offset_n;
  constexpr int increment = THREAD_N >= 4 ? 4 : THREAD_N;

  for (int warp_iter_m = 0; warp_iter_m < WARP_ITER_M; warp_iter_m++)
    for (int warp_iter_n = 0; warp_iter_n < WARP_ITER_N; warp_iter_n++)
      for (int m = 0; m < THREAD_M; m++)
        for (int n = 0; n < THREAD_N; n += increment) {
          float *tmp_addr = &acc[warp_iter_m][warp_iter_n][m][n];

          // TODO: handle n % 4 != 0
          const int row = warp_id_m * WARP_M + warp_iter_m * MMA_M + lane_id_m * THREAD_M + m;
          const int col = warp_id_n * WARP_N + warp_iter_n * MMA_N + lane_id_n * THREAD_N + n;

          if (row < (M - offset_m) && col < (N - offset_n)) {
            if constexpr (increment == 4)
              reinterpret_cast<float4 *>(&C[row * N + col])[0] = reinterpret_cast<float4 *>(tmp_addr)[0];
            if constexpr (increment == 2)
              reinterpret_cast<float2 *>(&C[row * N + col])[0] = reinterpret_cast<float2 *>(tmp_addr)[0];
            if constexpr (increment == 1)
              C[row * N + col] = tmp_addr[0];
          }
        }
}

void matmul_v5(const float *A, const float *B, float *C, int M, int N, int K) {
  const int BLOCK_SIZE = 256;
  const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 32;
  const int WARP_N = 64;  // WARP_M = 32
  const int MMA_M = 16, MMA_N = 8;
  const int THREAD_N = 2;  // THREAD_M = MMA_M * MMA_N / 32 / THREAD_N = 2
  const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
  matmul_v5_kernel<BLOCK_SIZE, BLOCK_M, BLOCK_N, BLOCK_K, WARP_N, MMA_M, MMA_N, THREAD_N><<<grid_size, BLOCK_SIZE>>>(A, B, C, M, N, K);
}
