#include "hip/hip_runtime.h"
#include "common.h"

#include <hip/hip_bf16.h>
#include <cstdint>
#include <float.h>
#include <iostream>

template<int BLOCK_Q, int BLOCK_KV, int DIM, int NUM_WARPS>
__launch_bounds__(NUM_WARPS * WARP_SIZE)
__global__
void attention_v4_kernel(
  const hip_bfloat16 *Q,  // [bs, len_q, DIM]
  const hip_bfloat16 *K,  // [bs, len_kv, DIM]
  const hip_bfloat16 *V,  // [bs, len_kv, DIM]
  hip_bfloat16 *O,        // [bs, len_q, DIM]
  int bs,
  int len_q,
  int len_kv) {

  constexpr int TB_SIZE = NUM_WARPS * WARP_SIZE;

  const int bid = blockIdx.x;
  const int tid = threadIdx.x;
  const int warp_id = tid / WARP_SIZE;
  const int lane_id = tid % WARP_SIZE;

  // each threadblock handles 1 BLOCK_Q
  const int num_q_blocks = cdiv(len_q, BLOCK_Q);
  const int bs_id = bid / num_q_blocks;
  const int q_block_id = bid % num_q_blocks;

  Q += (bs_id * num_q_blocks + q_block_id) * BLOCK_Q * DIM;
  K += bs_id * len_kv * DIM;
  V += bs_id * len_kv * DIM;
  O += (bs_id * num_q_blocks + q_block_id) * BLOCK_Q * DIM;

  // we overlap Q_shm with (K_shm + V_shm), since we only need to load Q_shm once
  extern __shared__ hip_bfloat16 shm[];
  const uint32_t Q_shm = __cvta_generic_to_shared(shm);
  const uint32_t K_shm = Q_shm;
  const uint32_t V_shm = K_shm + BLOCK_KV * DIM * sizeof(hip_bfloat16);

  // FA2: shard BLOCK_Q among all warps
  // replicate K and V on all warps
  constexpr int WARP_Q = BLOCK_Q / NUM_WARPS;

  // mma.m16n8k16
  constexpr int MMA_M = 16;
  constexpr int MMA_N = 8;
  constexpr int MMA_K = 16;
  constexpr int num_A_regs = MMA_M * MMA_K * sizeof(hip_bfloat16) / 4 / WARP_SIZE;
  constexpr int num_B_regs = MMA_N * MMA_K * sizeof(hip_bfloat16) / 4 / WARP_SIZE;
  constexpr int num_acc_regs = MMA_M * MMA_N / WARP_SIZE;

  // set up registers
  uint32_t Q_regs[WARP_Q / MMA_M][DIM / MMA_K][num_A_regs];
  uint32_t K_regs[BLOCK_KV / MMA_N][DIM / MMA_K][num_B_regs];

  // let compiler decide register reuse?
  uint32_t P_regs[WARP_Q / MMA_M][BLOCK_KV / MMA_K][num_A_regs];
  uint32_t V_regs[BLOCK_KV / MMA_K][DIM / MMA_N][num_B_regs];

  // we use the same registers for O_regs and PV_regs
  // rescale O_regs once we obtain new rowmax, then accumulate to O_regs
  float O_regs[WARP_Q / MMA_M][DIM / MMA_N][num_acc_regs] = {};

  // pre-compute address and swizzling for ldmatrix
  uint32_t Q_shm_thread, K_shm_thread, V_shm_thread;
  {
    // A tile
    const int row_off = warp_id * WARP_Q + (lane_id % 16);
    const int col_off = lane_id / 16 * 8;
    Q_shm_thread = swizzle<DIM * sizeof(hip_bfloat16)>(Q_shm + (row_off * DIM + col_off) * sizeof(hip_bfloat16));
  }
  {
    // B tile
    const int row_off = lane_id % 8;
    const int col_off = lane_id / 8 * 8;
    K_shm_thread = swizzle<DIM * sizeof(hip_bfloat16)>(K_shm + (row_off * DIM + col_off) * sizeof(hip_bfloat16));
  }
  {
    // B tile trans
    const int row_off = lane_id % 16;
    const int col_off = lane_id / 16 * 8;
    V_shm_thread = swizzle<DIM * sizeof(hip_bfloat16)>(V_shm + (row_off * DIM + col_off) * sizeof(hip_bfloat16));
  }

  const float softmax_scale = rsqrtf(static_cast<float>(DIM));

  float rowmax[WARP_Q / MMA_M][2];
  float rowsumexp[WARP_Q / MMA_M][2] = {};

  for (int mma_id_q = 0; mma_id_q < WARP_Q / MMA_M; mma_id_q++) {
    rowmax[mma_id_q][0] = -FLT_MAX;
    rowmax[mma_id_q][1] = -FLT_MAX;
  }

  // load Q [BLOCK_Q, DIM]
  global_to_shared_swizzle<BLOCK_Q, DIM, TB_SIZE>(Q_shm, Q, DIM, tid);
  asm volatile("cp.async.commit_group;");
  asm volatile("cp.async.wait_all;");
  __syncthreads();

  // shared -> registers
  for (int mma_id_q = 0; mma_id_q < WARP_Q / MMA_M; mma_id_q++)
    for (int mma_id_d = 0; mma_id_d < DIM / MMA_K; mma_id_d++) {
      uint32_t addr = Q_shm_thread;
      addr += mma_id_q * MMA_M * DIM * sizeof(hip_bfloat16);  // row
      addr ^= mma_id_d * MMA_K * sizeof(hip_bfloat16);  // col
      ldmatrix_x4(Q_regs[mma_id_q][mma_id_d], addr);
    }
  // we need a syncthreads() here so that we don't load K global->shared
  // before finishing loading Q shared->reg
  __syncthreads();

  const int num_kv_iter = cdiv(len_kv, BLOCK_KV);

  auto load_K = [&](int kv_id) {
    if (kv_id < num_kv_iter) {
      const uint32_t dst = K_shm + (kv_id % 2) * (2 * BLOCK_KV * DIM * sizeof(hip_bfloat16));
      global_to_shared_swizzle<BLOCK_KV, DIM, TB_SIZE>(dst, K, DIM, tid);
      K += BLOCK_KV * DIM;
    }
    asm volatile("cp.async.commit_group;");
  };
  auto load_V = [&](int kv_id) {
    if (kv_id < num_kv_iter) {
      const uint32_t dst = V_shm + (kv_id % 2) * (2 * BLOCK_KV * DIM * sizeof(hip_bfloat16));
      global_to_shared_swizzle<BLOCK_KV, DIM, TB_SIZE>(dst, V, DIM, tid);
      V += BLOCK_KV * DIM;
    }
    asm volatile("cp.async.commit_group;");
  };

  // prefetch K and V
  load_K(0);
  load_V(0);

  for (int kv_id = 0; kv_id < num_kv_iter; kv_id++) {
    float QK_regs[WARP_Q / MMA_M][BLOCK_KV / MMA_N][num_acc_regs] = {};

    // prefetch K
    load_K(kv_id + 1);
    asm volatile("cp.async.wait_group 2;");
    __syncthreads();

    // shared -> registers
    for (int mma_id_kv = 0; mma_id_kv < BLOCK_KV / MMA_N; mma_id_kv++)
      for (int mma_id_d = 0; mma_id_d < DIM / MMA_K; mma_id_d += 2) {
        uint32_t addr = K_shm_thread + (kv_id % 2) * (2 * BLOCK_KV * DIM * sizeof(hip_bfloat16));
        addr += mma_id_kv * MMA_N * DIM * sizeof(hip_bfloat16);  // row
        addr ^= mma_id_d * MMA_K * sizeof(hip_bfloat16);  // col
        ldmatrix_x4(K_regs[mma_id_kv][mma_id_d], addr);
      }

    // MMA S = Q @ K.T [BLOCK_Q, BLOCK_KV]
    for (int mma_id_q = 0; mma_id_q < WARP_Q / MMA_M; mma_id_q++)
      for (int mma_id_kv = 0; mma_id_kv < BLOCK_KV / MMA_N; mma_id_kv++)
        for (int mma_id_d = 0; mma_id_d < DIM / MMA_K; mma_id_d++)
          mma_m16n8k16(Q_regs[mma_id_q][mma_id_d],
                       K_regs[mma_id_kv][mma_id_d],
                       QK_regs[mma_id_q][mma_id_kv]);

    // prefetch V
    load_V(kv_id + 1);

    for (int mma_id_q = 0; mma_id_q < WARP_Q / MMA_M; mma_id_q++) {
      // apply softmax scale
      for (int mma_id_kv = 0; mma_id_kv < BLOCK_KV / MMA_N; mma_id_kv++)
        for (int reg_id = 0; reg_id < num_acc_regs; reg_id++)
          QK_regs[mma_id_q][mma_id_kv][reg_id] *= softmax_scale;

      // rowmax
      float this_rowmax[2] = {-FLT_MAX, -FLT_MAX};
      for (int mma_id_kv = 0; mma_id_kv < BLOCK_KV / MMA_N; mma_id_kv++) {
        float *regs = QK_regs[mma_id_q][mma_id_kv];
        this_rowmax[0] = max(this_rowmax[0], max(regs[0], regs[1]));  // c0 and c1
        this_rowmax[1] = max(this_rowmax[1], max(regs[2], regs[3]));  // c2 and c3
      }

      // butterfly reduction within 4 threads
      this_rowmax[0] = max(this_rowmax[0], __shfl_xor_sync(0xFFFF'FFFF, this_rowmax[0], 1));
      this_rowmax[0] = max(this_rowmax[0], __shfl_xor_sync(0xFFFF'FFFF, this_rowmax[0], 2));
      this_rowmax[1] = max(this_rowmax[1], __shfl_xor_sync(0xFFFF'FFFF, this_rowmax[1], 1));
      this_rowmax[1] = max(this_rowmax[1], __shfl_xor_sync(0xFFFF'FFFF, this_rowmax[1], 2));

      // new rowmax
      this_rowmax[0] = max(this_rowmax[0], rowmax[mma_id_q][0]);
      this_rowmax[1] = max(this_rowmax[1], rowmax[mma_id_q][1]);

      // rescale for previous O
      float rescale[2];
      rescale[0] = __expf(rowmax[mma_id_q][0] - this_rowmax[0]);
      rescale[1] = __expf(rowmax[mma_id_q][1] - this_rowmax[1]);
      for (int mma_id_d = 0; mma_id_d < DIM / MMA_N; mma_id_d++) {
        O_regs[mma_id_q][mma_id_d][0] *= rescale[0];
        O_regs[mma_id_q][mma_id_d][1] *= rescale[0];
        O_regs[mma_id_q][mma_id_d][2] *= rescale[1];
        O_regs[mma_id_q][mma_id_d][3] *= rescale[1];
      }

      // save new rowmax
      rowmax[mma_id_q][0] = this_rowmax[0];
      rowmax[mma_id_q][1] = this_rowmax[1];

      // rowsumexp
      float this_rowsumexp[2] = {};
      for (int mma_id_kv = 0; mma_id_kv < BLOCK_KV / MMA_N; mma_id_kv++) {
        float *regs = QK_regs[mma_id_q][mma_id_kv];
        regs[0] = __expf(regs[0] - rowmax[mma_id_q][0]);  // c0
        regs[1] = __expf(regs[1] - rowmax[mma_id_q][0]);  // c1
        regs[2] = __expf(regs[2] - rowmax[mma_id_q][1]);  // c2
        regs[3] = __expf(regs[3] - rowmax[mma_id_q][1]);  // c3

        this_rowsumexp[0] += regs[0] + regs[1];
        this_rowsumexp[1] += regs[2] + regs[3];

        // pack to P registers for next MMA
        // we need to change from m16n8 to m16k16
        nv_bfloat162 *this_P_regs = reinterpret_cast<nv_bfloat162 *>(P_regs[mma_id_q][mma_id_kv / 2]);
        this_P_regs[(mma_id_kv % 2) * 2] = __float22bfloat162_rn({regs[0], regs[1]});
        this_P_regs[(mma_id_kv % 2) * 2 + 1] = __float22bfloat162_rn({regs[2], regs[3]});
      }

      // butterfly reduction within 4 threads
      this_rowsumexp[0] += __shfl_xor_sync(0xFFFF'FFFF, this_rowsumexp[0], 1);
      this_rowsumexp[0] += __shfl_xor_sync(0xFFFF'FFFF, this_rowsumexp[0], 2);
      this_rowsumexp[1] += __shfl_xor_sync(0xFFFF'FFFF, this_rowsumexp[1], 1);
      this_rowsumexp[1] += __shfl_xor_sync(0xFFFF'FFFF, this_rowsumexp[1], 2);

      // accumulate to total rowsumexp
      rowsumexp[mma_id_q][0] = rowsumexp[mma_id_q][0] * rescale[0] + this_rowsumexp[0];
      rowsumexp[mma_id_q][1] = rowsumexp[mma_id_q][1] * rescale[1] + this_rowsumexp[1];
    }

    // wait V load to finish
    asm volatile("cp.async.wait_group 2;");
    __syncthreads();

    // shared -> registers
    for (int mma_id_kv = 0; mma_id_kv < BLOCK_KV / MMA_K; mma_id_kv++)
      for (int mma_id_d = 0; mma_id_d < DIM / MMA_N; mma_id_d += 2) {
        uint32_t addr = V_shm_thread + (kv_id % 2) * (2 * BLOCK_KV * DIM * sizeof(hip_bfloat16));
        addr += mma_id_kv * MMA_K * DIM * sizeof(hip_bfloat16);  // row
        addr ^= mma_id_d * MMA_N * sizeof(hip_bfloat16);  // col
        ldmatrix_x4_trans(V_regs[mma_id_kv][mma_id_d], addr);
      }

    // MMA P = S @ V [BLOCK_Q, DIM]
    for (int mma_id_q = 0; mma_id_q < WARP_Q / MMA_M; mma_id_q++)
      for (int mma_id_d = 0; mma_id_d < DIM / MMA_N; mma_id_d++)
        for (int mma_id_kv = 0; mma_id_kv < BLOCK_KV / MMA_K; mma_id_kv++)
          mma_m16n8k16(P_regs[mma_id_q][mma_id_kv],
                       V_regs[mma_id_kv][mma_id_d],
                       O_regs[mma_id_q][mma_id_d]);
  }

  // write to O
  for (int mma_id_q = 0; mma_id_q < WARP_Q / MMA_M; mma_id_q++)
    for (int mma_id_d = 0; mma_id_d < DIM / MMA_N; mma_id_d++) {
      const int row = warp_id * WARP_Q + mma_id_q * MMA_M + (lane_id / 4);
      const int col = mma_id_d * MMA_N + (lane_id % 4) * 2;
      hip_bfloat16 *O_ptr = O + row * DIM + col;

      // divide by softmax denominator
      float *this_O_regs = O_regs[mma_id_q][mma_id_d];
      this_O_regs[0] /= rowsumexp[mma_id_q][0];
      this_O_regs[1] /= rowsumexp[mma_id_q][0];
      this_O_regs[2] /= rowsumexp[mma_id_q][1];
      this_O_regs[3] /= rowsumexp[mma_id_q][1];

      reinterpret_cast<nv_bfloat162 *>(O_ptr)[0] = __float22bfloat162_rn({this_O_regs[0], this_O_regs[1]});
      reinterpret_cast<nv_bfloat162 *>(O_ptr + 8 * DIM)[0] = __float22bfloat162_rn({this_O_regs[2], this_O_regs[3]});
    }
}

void attention_v4(
  const hip_bfloat16 *Q,  // [bs, len_q, DIM]
  const hip_bfloat16 *K,  // [bs, len_kv, DIM]
  const hip_bfloat16 *V,  // [bs, len_kv, DIM]
  hip_bfloat16 *O,        // [bs, len_q, DIM]
  int bs,
  int len_q,
  int len_kv,
  int dim) {

  if (dim != 128) {
    std::cerr << "Unsupported dim=" << dim << std::endl;
    exit(1);
  }

  const int BLOCK_Q = 64;
  const int BLOCK_KV = 32;
  const int DIM = 128;
  const int NUM_WARPS = 4;

  const int num_blocks = bs * cdiv(len_q, BLOCK_Q);
  const int TB_SIZE = NUM_WARPS * WARP_SIZE;
  const int shm_size = max(BLOCK_Q, BLOCK_KV * 2 * 2) * DIM * sizeof(hip_bfloat16);

  auto kernel = attention_v4_kernel<BLOCK_Q, BLOCK_KV, DIM, NUM_WARPS>;
  launch_kernel(kernel, num_blocks, TB_SIZE, shm_size, Q, K, V, O, bs, len_q, len_kv);
}
