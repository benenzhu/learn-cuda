#include "hip/hip_runtime.h"
#include "common.h"
#include <assert.h>
#include <cmath>
#include <cstdint>
#include <hip/hip_bf16.h>
#include <stdio.h>

#define CUDA_CHECK(call)                                                                                               \
  do {                                                                                                                 \
    hipError_t err = call;                                                                                            \
    if (err != hipSuccess) {                                                                                          \
      fprintf(stderr, "CUDA error %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__);                        \
      exit(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  } while (0)

__host__ __device__
constexpr int cdiv(int a, int b) { return (a + b - 1) / b; }
constexpr bool is_power_of_two(int x) { return x > 0 && (x & (x - 1)) == 0; } // https://stackoverflow.com/a/1804686
constexpr int WARP_SIZE = 32;

// convert generic address (C++ address, 64-bit) to shared state space address (32-bit)
// all PTX instructions expect share memory address to be in shared state space (not 100%)
__device__
uint32_t cvta_shared(const void *ptr) { return static_cast<uint32_t>(__cvta_generic_to_shared(ptr)); }

// https://docs.nvidia.com/cuda/parallel-thread-execution/#data-movement-and-conversion-instructions-non-bulk-copy
__device__
void cp_async(uint32_t dst, const void *src) {
  // .ca means cache to L1 and L2. .cg means cache to L2 only.
  // .cg only accepts cp-size=16
  // .ca results in significantly slower kernel, probably because it uses up L1 resources
  // + additional copy, which is unnecessary, since we already manually cache it in shared memory.
  asm volatile("cp.async.cg.shared.global [%0], [%1], 16;" ::"r"(dst), "l"(src));
};

__device__
void cp_async_commit_group() { asm volatile("cp.async.commit_group;"); };

template <int N>
__device__
void cp_async_wait_group() { asm volatile("cp.async.wait_group %0;" ::"n"(N)); };

__device__
void cp_async_wait_all() { asm volatile("cp.async.wait_all;"); };

// NOTE: stride in bytes
template <int STRIDE>
__device__
uint32_t swizzle(uint32_t index) {
  // no need swizzling
  if constexpr (STRIDE == 16)
    return index;

  uint32_t row_idx = (index / STRIDE) % 8;
  uint32_t bits_to_xor = row_idx / max(64 / STRIDE, 1);
  return index ^ (bits_to_xor << 4);
}

template <int TB_SIZE, int HEIGHT, int WIDTH, int OUT_STRIDE>
__device__
void global_to_shared(const hip_bfloat16 *in, int in_stride, hip_bfloat16 *out, int tid) {
  // number of elements to do 128-bit/16-byte load
  // e.g. FP32 -> 4 elements, BF16 -> 8 elements.
  using TLoad = uint4;
  constexpr int num_elems = sizeof(TLoad) / sizeof(hip_bfloat16);

  // NOTE: write loop this way to make sure the compiler can fully unroll it.
  constexpr int num_iters = (HEIGHT * WIDTH) / (TB_SIZE * num_elems);
  for (int iter = 0; iter < num_iters; iter++) {
    const int idx = (iter * TB_SIZE + tid) * num_elems;
    const int row = idx / WIDTH;
    const int col = idx % WIDTH;
    TLoad tmp = reinterpret_cast<const TLoad *>(in + row * in_stride + col)[0];
    reinterpret_cast<TLoad *>(out + row * OUT_STRIDE + col)[0] = tmp;
  }
}

template <int TB_SIZE, int HEIGHT, int WIDTH, int OUT_STRIDE, bool use_swizzle>
__device__
void global_to_shared_async(const hip_bfloat16 *in, int in_stride, hip_bfloat16 *out, int tid) {
  constexpr int num_elems = 16 / sizeof(hip_bfloat16);  // cp.async cp-size = 16

  // convert to shared state space outside of the loop
  // TODO: move this to kernel body
  uint32_t out_addr = cvta_shared(out);

  constexpr int num_iters = (HEIGHT * WIDTH) / (TB_SIZE * num_elems);
  for (int iter = 0; iter < num_iters; iter++) {
    const int idx = (iter * TB_SIZE + tid) * num_elems;
    const int row = idx / WIDTH;
    const int col = idx % WIDTH;

    uint32_t dst_addr = out_addr + (row * OUT_STRIDE + col) * sizeof(hip_bfloat16);
    if constexpr (use_swizzle)
      dst_addr = swizzle<OUT_STRIDE * sizeof(hip_bfloat16)>(dst_addr);
    cp_async(dst_addr, in + row * in_stride + col);
  }
}

template <typename T, typename... Args>
void launch_kernel(T *kernel, int num_blocks, int block_size, int shm_size, Args... args) {
  if (shm_size > 48'000)
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shm_size));

  kernel<<<num_blocks, block_size, shm_size>>>(args...);
  CUDA_CHECK(hipGetLastError());
}

template <int BLOCK_M, int BLOCK_N, int BLOCK_K, int NUM_WARP_M, int NUM_WARP_N, int SHM_STRIDE, bool use_cp_async,
          bool use_swizzle>
__launch_bounds__(NUM_WARP_M * NUM_WARP_N * WARP_SIZE) // maxThreadsPerBlock
__global__
void matmul_v1_kernel(const hip_bfloat16 *A, const hip_bfloat16 *B, hip_bfloat16 *C, int M, int N, int K) {
  constexpr int MMA_M = 16;
  constexpr int MMA_N = 8;
  constexpr int MMA_K = 16;
  static_assert(BLOCK_M % NUM_WARP_M == 0);
  static_assert(BLOCK_N % NUM_WARP_N == 0);
  static_assert(BLOCK_K % MMA_K == 0);
  constexpr int WARP_M = BLOCK_M / NUM_WARP_M;
  constexpr int WARP_N = BLOCK_N / NUM_WARP_N;
  static_assert(WARP_M % MMA_M == 0);
  static_assert(WARP_N % MMA_N == 0);
  static_assert(use_cp_async || !use_swizzle); // use_swizzle=true requires use_cp_async=true
  constexpr int TB_SIZE = NUM_WARP_M * NUM_WARP_N * WARP_SIZE;

  // each warp will do (NUM_MMA_M * NUM_MMA_N) MMAs
  constexpr int NUM_MMA_M = WARP_M / MMA_M;
  constexpr int NUM_MMA_N = WARP_N / MMA_N;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int warp_id = tid / WARP_SIZE;
  const int lane_id = tid % WARP_SIZE;

  // TODO: threadblock swizzling to improve L2 cache hit rate
  const int num_blocks_n = cdiv(N, BLOCK_N);
  const int bid_m = bid / num_blocks_n;
  const int bid_n = bid % num_blocks_n;
  const int offset_m = bid_m * BLOCK_M;
  const int offset_n = bid_n * BLOCK_N;

  const int warp_id_m = warp_id / NUM_WARP_N;
  const int warp_id_n = warp_id % NUM_WARP_N;

  // A is row-major, B is column-major, C is row-major
  A += offset_m * K;
  B += offset_n * K;
  C += (offset_m + warp_id_m * WARP_M) * N + (offset_n + warp_id_n * WARP_N);

  extern __shared__ hip_bfloat16 shm[];
  hip_bfloat16 *A_shared = shm;                               // BLOCK_M * BLOCK_K
  hip_bfloat16 *B_shared = A_shared + (BLOCK_M * SHM_STRIDE); // BLOCK_N * BLOCK_K

  // all registers are 32-bit (4-byte)
  // - we accumulate to FP32, which is exactly 32-bit
  // - our inputs are FP16/BF16, hence each register holds 2 elements
  // - inputs and accumulate are distributed across 32 threads in a warp
  // for m16n8k8, each thread holds
  // - 4 output float
  // - 4 input A FP16/BF16
  // - 2 input B FP16/BF16
  constexpr int num_acc_regs = MMA_M * MMA_N / WARP_SIZE;
  constexpr int num_A_regs = MMA_M * MMA_K * sizeof(hip_bfloat16) / 4 / WARP_SIZE;
  constexpr int num_B_regs = MMA_N * MMA_K * sizeof(hip_bfloat16) / 4 / WARP_SIZE;
  float acc[NUM_MMA_M][NUM_MMA_N][num_acc_regs] = {};

  for (int block_k = 0; block_k < K; block_k += BLOCK_K) {
    if constexpr (use_cp_async) {
      global_to_shared_async<TB_SIZE, BLOCK_M, BLOCK_K, SHM_STRIDE, use_swizzle>(A, K, A_shared, tid);
      global_to_shared_async<TB_SIZE, BLOCK_N, BLOCK_K, SHM_STRIDE, use_swizzle>(B, K, B_shared, tid);
      cp_async_wait_all();
    } else {
      global_to_shared<TB_SIZE, BLOCK_M, BLOCK_K, SHM_STRIDE>(A, K, A_shared, tid);
      global_to_shared<TB_SIZE, BLOCK_N, BLOCK_K, SHM_STRIDE>(B, K, B_shared, tid);
    }
    __syncthreads();

    for (int mma_k = 0; mma_k < BLOCK_K; mma_k += MMA_K) {
      // for m16n8k8
      // https://docs.nvidia.com/cuda/parallel-thread-execution/#warp-level-matrix-fragment-mma-1688
      //   A\B   [8x8-0]
      // [8x8-0]
      // [8x8-1]
      // where each [8x8] matrix can be loaded from shared memory with ldmatrix
      // https://docs.nvidia.com/cuda/parallel-thread-execution/#warp-level-matrix-instructions-ldmatrix

      // for m16n8k16
      // https://docs.nvidia.com/cuda/parallel-thread-execution/#warp-level-matrix-fragment-mma-16816-float
      //                [8x8-0]
      //       A\B      [8x8-1]
      // [8x8-0][8x8-2]
      // [8x8-1][8x8-3]

      // select the tile this warp is responsible for
      const hip_bfloat16 *A_shm_warp = A_shared + (warp_id_m * WARP_M) * SHM_STRIDE + mma_k;
      const hip_bfloat16 *B_shm_warp = B_shared + (warp_id_n * WARP_N) * SHM_STRIDE + mma_k;

      // to use ldmatrix: each thread holds the address of 1 row e.g.
      // - thread 0 holds address of row 0
      // - thread 1 holds address of row 1, and so on
      // when loading multiple matrices, thread0-7 specifies the 1st matrix,
      // thread 8-15 specifies the 2nd matrix, and so on

      // load B to registers
      uint32_t B_reg[NUM_MMA_N][num_B_regs];
      for (int mma_id_n = 0; mma_id_n < NUM_MMA_N; mma_id_n++) {
        // NOTE: we can reduce unnecessary address calculation if we know MMA_K=8 or 16
        // convert generic address to .shared state space address expected by inline PTX
        const hip_bfloat16 *B_ptr = B_shm_warp + (mma_id_n * MMA_N + (lane_id % 8)) * SHM_STRIDE + (lane_id / 8) * 8;
        uint32_t B_addr = cvta_shared(B_ptr);
        if constexpr (use_swizzle)
          B_addr = swizzle<SHM_STRIDE * sizeof(hip_bfloat16)>(B_addr);
        ldmatrix<num_B_regs>(B_reg[mma_id_n], B_addr);
      }

      for (int mma_id_m = 0; mma_id_m < NUM_MMA_M; mma_id_m++) {
        // load A to registers
        uint32_t A_reg[num_A_regs];
        const hip_bfloat16 *A_ptr = A_shm_warp + (mma_id_m * MMA_M + (lane_id % 16)) * SHM_STRIDE + (lane_id / 16) * 8;
        uint32_t A_addr = cvta_shared(A_ptr);
        if constexpr (use_swizzle)
          A_addr = swizzle<SHM_STRIDE * sizeof(hip_bfloat16)>(A_addr);
        ldmatrix<num_A_regs>(A_reg, A_addr);

        // call mma
        for (int mma_id_n = 0; mma_id_n < NUM_MMA_N; mma_id_n++)
          mma(A_reg, B_reg[mma_id_n], acc[mma_id_m][mma_id_n]);
      }
    }
    __syncthreads();

    A += BLOCK_K;
    B += BLOCK_K;
  }

  // check output layout here
  // https://docs.nvidia.com/cuda/parallel-thread-execution/#mma-1688-c-f16-f32
  // m16n8k16 has the same layout
  const int a0_row = lane_id / 4;
  const int a0_col = (lane_id % 4) * 2;
  C += a0_row * N + a0_col;

  // NOTE: we can do some warp shuffle to get coalesced write
  for (int mma_id_m = 0; mma_id_m < NUM_MMA_M; mma_id_m++)
    for (int mma_id_n = 0; mma_id_n < NUM_MMA_N; mma_id_n++) {
      hip_bfloat16 *C_local = C + (mma_id_m * MMA_M) * N + (mma_id_n * MMA_N);
      float *regs = acc[mma_id_m][mma_id_n];

      reinterpret_cast<nv_bfloat162 *>(C_local)[0]         = __float22bfloat162_rn({regs[0], regs[1]});  // c0 and c1
      reinterpret_cast<nv_bfloat162 *>(C_local + 8 * N)[0] = __float22bfloat162_rn({regs[2], regs[3]});  // c2 and c3
    }
}

void matmul_v1(const hip_bfloat16 *A, const hip_bfloat16 *B, hip_bfloat16 *C, int M, int N, int K) {
  assert(is_power_of_two(M) && "M must be a power of 2");
  assert(is_power_of_two(N) && "N must be a power of 2");
  assert(is_power_of_two(K) && "K must be a power of 2");

  // 4 warps
  const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 64;
  const int NUM_WARP_M = 2, NUM_WARP_N = 2;
  const int SHM_STRIDE = BLOCK_K; // no padding
  const int use_cp_async = false;
  const int use_swizzle = false;

  auto kernel =
      matmul_v1_kernel<BLOCK_M, BLOCK_N, BLOCK_K, NUM_WARP_M, NUM_WARP_N, SHM_STRIDE, use_cp_async, use_swizzle>;

  const int TB_SIZE = NUM_WARP_M * NUM_WARP_N * WARP_SIZE;
  const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
  const int shm_size = (BLOCK_M + BLOCK_N) * SHM_STRIDE * sizeof(hip_bfloat16);

  launch_kernel(kernel, grid_size, TB_SIZE, shm_size, A, B, C, M, N, K);
}

void matmul_v2(const hip_bfloat16 *A, const hip_bfloat16 *B, hip_bfloat16 *C, int M, int N, int K) {
  assert(is_power_of_two(M) && "M must be a power of 2");
  assert(is_power_of_two(N) && "N must be a power of 2");
  assert(is_power_of_two(K) && "K must be a power of 2");

  // 4 warps
  const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 64;
  const int NUM_WARP_M = 2, NUM_WARP_N = 2;
  const int SHM_STRIDE = BLOCK_K; // no padding
  const int use_cp_async = true;
  const int use_swizzle = false;

  auto kernel =
      matmul_v1_kernel<BLOCK_M, BLOCK_N, BLOCK_K, NUM_WARP_M, NUM_WARP_N, SHM_STRIDE, use_cp_async, use_swizzle>;

  const int TB_SIZE = NUM_WARP_M * NUM_WARP_N * WARP_SIZE;
  const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
  const int shm_size = (BLOCK_M + BLOCK_N) * SHM_STRIDE * sizeof(hip_bfloat16);

  launch_kernel(kernel, grid_size, TB_SIZE, shm_size, A, B, C, M, N, K);
}

void matmul_v3(const hip_bfloat16 *A, const hip_bfloat16 *B, hip_bfloat16 *C, int M, int N, int K) {
  assert(is_power_of_two(M) && "M must be a power of 2");
  assert(is_power_of_two(N) && "N must be a power of 2");
  assert(is_power_of_two(K) && "K must be a power of 2");

  // 4 warps
  const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 64;
  const int NUM_WARP_M = 2, NUM_WARP_N = 2;
  const int SHM_STRIDE = BLOCK_K + 8; // pad shmem to avoid bank conflict
  const int use_cp_async = true;
  const int use_swizzle = false;

  auto kernel =
      matmul_v1_kernel<BLOCK_M, BLOCK_N, BLOCK_K, NUM_WARP_M, NUM_WARP_N, SHM_STRIDE, use_cp_async, use_swizzle>;

  const int TB_SIZE = NUM_WARP_M * NUM_WARP_N * WARP_SIZE;
  const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
  const int shm_size = (BLOCK_M + BLOCK_N) * SHM_STRIDE * sizeof(hip_bfloat16);

  launch_kernel(kernel, grid_size, TB_SIZE, shm_size, A, B, C, M, N, K);
}

void matmul_v4(const hip_bfloat16 *A, const hip_bfloat16 *B, hip_bfloat16 *C, int M, int N, int K) {
  assert(is_power_of_two(M) && "M must be a power of 2");
  assert(is_power_of_two(N) && "N must be a power of 2");
  assert(is_power_of_two(K) && "K must be a power of 2");

  // 4 warps
  const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 64;
  const int NUM_WARP_M = 2, NUM_WARP_N = 2;
  const int SHM_STRIDE = BLOCK_K;
  const int use_cp_async = true;
  const int use_swizzle = true;

  auto kernel =
      matmul_v1_kernel<BLOCK_M, BLOCK_N, BLOCK_K, NUM_WARP_M, NUM_WARP_N, SHM_STRIDE, use_cp_async, use_swizzle>;

  const int TB_SIZE = NUM_WARP_M * NUM_WARP_N * WARP_SIZE;
  const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
  const int shm_size = (BLOCK_M + BLOCK_N) * SHM_STRIDE * sizeof(hip_bfloat16);

  launch_kernel(kernel, grid_size, TB_SIZE, shm_size, A, B, C, M, N, K);
}

template <int TB_SIZE, int HEIGHT, int WIDTH>
__device__
void global_to_shared_async(const hip_bfloat16 *in, int in_stride, uint32_t out, int tid) {
  constexpr int num_elems = 16 / sizeof(hip_bfloat16);
  constexpr int num_iters = (HEIGHT * WIDTH) / (TB_SIZE * num_elems);

  for (int iter = 0; iter < num_iters; iter++) {
    const int idx = (iter * TB_SIZE + tid) * num_elems;
    const int row = idx / WIDTH;
    const int col = idx % WIDTH;

    // NOTE: perhaps we can move swizzle out of this loop as well
    uint32_t dst_addr = swizzle<WIDTH * sizeof(hip_bfloat16)>(out + (row * WIDTH + col) * sizeof(hip_bfloat16));
    cp_async(dst_addr, in + row * in_stride + col);
  }
}

template <int BLOCK_M, int BLOCK_N, int BLOCK_K, int NUM_WARP_M, int NUM_WARP_N>
__launch_bounds__(NUM_WARP_M * NUM_WARP_N * WARP_SIZE) // maxThreadsPerBlock
__global__
void matmul_v5_kernel(const hip_bfloat16 *A, const hip_bfloat16 *B, hip_bfloat16 *C, int M, int N, int K) {
  constexpr int MMA_M = 16;
  constexpr int MMA_N = 16;
  constexpr int MMA_K = 16;
  static_assert(BLOCK_M % NUM_WARP_M == 0);
  static_assert(BLOCK_N % NUM_WARP_N == 0);
  static_assert(BLOCK_K % MMA_K == 0);
  constexpr int WARP_M = BLOCK_M / NUM_WARP_M;
  constexpr int WARP_N = BLOCK_N / NUM_WARP_N;
  static_assert(WARP_M % MMA_M == 0);
  static_assert(WARP_N % MMA_N == 0);
  constexpr int TB_SIZE = NUM_WARP_M * NUM_WARP_N * WARP_SIZE;
  constexpr int NUM_MMA_M = WARP_M / MMA_M;
  constexpr int NUM_MMA_N = WARP_N / MMA_N;
  constexpr int NUM_MMA_K = BLOCK_K / MMA_K;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int warp_id = tid / WARP_SIZE;
  const int lane_id = tid % WARP_SIZE;

  // TODO: threadblock swizzling to improve L2 cache hit rate
  const int num_blocks_n = cdiv(N, BLOCK_N);
  const int bid_m = bid / num_blocks_n;
  const int bid_n = bid % num_blocks_n;
  const int offset_m = bid_m * BLOCK_M;
  const int offset_n = bid_n * BLOCK_N;

  const int warp_id_m = warp_id / NUM_WARP_N;
  const int warp_id_n = warp_id % NUM_WARP_N;

  // A is row-major, B is column-major, C is row-major
  A += offset_m * K;
  B += offset_n * K;
  C += (offset_m + warp_id_m * WARP_M) * N + (offset_n + warp_id_n * WARP_N);

  // convert shared memory address to 32-bit from the start
  extern __shared__ hip_bfloat16 shm[];
  const uint32_t A_shared = cvta_shared(shm);                                     // BLOCK_M * BLOCK_K
  const uint32_t B_shared = A_shared + (BLOCK_M * BLOCK_K) * sizeof(hip_bfloat16); // BLOCK_N * BLOCK_K

  constexpr int num_acc_regs = MMA_M * MMA_N / WARP_SIZE;
  constexpr int num_A_regs = MMA_M * MMA_K * sizeof(hip_bfloat16) / 4 / WARP_SIZE; // 4
  constexpr int num_B_regs = MMA_N * MMA_K * sizeof(hip_bfloat16) / 4 / WARP_SIZE; // 4
  float acc[NUM_MMA_M][NUM_MMA_N][num_acc_regs] = {};

  // pre-compute address used for ldmatrix
  // also pre-compute swizzling
  const int A_offm = (warp_id_m * WARP_M) + (lane_id % 16);
  const int A_offk = (lane_id / 16) * 8;
  const uint32_t A_shm_thread = swizzle<BLOCK_K * sizeof(hip_bfloat16)>(A_shared + (A_offm * BLOCK_K + A_offk) * sizeof(hip_bfloat16));

  const int B_offn = (warp_id_n * WARP_N) + (lane_id % 8) + (lane_id / 16) * 8;
  const int B_offk = ((lane_id % 16) / 8) * 8;
  const uint32_t B_shm_thread = swizzle<BLOCK_K * sizeof(hip_bfloat16)>(B_shared + (B_offn * BLOCK_K + B_offk) * sizeof(hip_bfloat16));

  for (int block_k = 0; block_k < K; block_k += BLOCK_K) {
    global_to_shared_async<TB_SIZE, BLOCK_M, BLOCK_K>(A, K, A_shared, tid);
    global_to_shared_async<TB_SIZE, BLOCK_N, BLOCK_K>(B, K, B_shared, tid);
    cp_async_wait_all();
    __syncthreads();

    for (int mma_id_k = 0; mma_id_k < NUM_MMA_K; mma_id_k++) {
      // iterate MMA_K=16 -> increment bit5 (32 bytes) -> affects swizzled bits
      // assume we have alignment (bit0-6 are all zeros), increment bit5
      // is equivalent to XOR mma_id_k directly, which is commutative with swizzling
      // -> we can move swizzling outside of this loop
      // the kernel compiles to fewer instructions, but no speedup

      // load B to registers
      uint32_t B_reg[NUM_MMA_N][num_B_regs];
      for (int mma_id_n = 0; mma_id_n < NUM_MMA_N; mma_id_n++) {
        const uint32_t B_addr = B_shm_thread + mma_id_n * MMA_N * BLOCK_K * sizeof(hip_bfloat16);
        ldmatrix<num_B_regs>(B_reg[mma_id_n], B_addr ^ (mma_id_k * MMA_K * sizeof(hip_bfloat16)));
      }

      for (int mma_id_m = 0; mma_id_m < NUM_MMA_M; mma_id_m++) {
        // load A to registers
        uint32_t A_reg[num_A_regs];
        const uint32_t A_addr = A_shm_thread + mma_id_m * MMA_M * BLOCK_K * sizeof(hip_bfloat16);
        ldmatrix<num_A_regs>(A_reg, A_addr ^ (mma_id_k * MMA_K * sizeof(hip_bfloat16)));

        // call mma
        for (int mma_id_n = 0; mma_id_n < NUM_MMA_N; mma_id_n++) {
          mma(A_reg, B_reg[mma_id_n], acc[mma_id_m][mma_id_n]);
          mma(A_reg, B_reg[mma_id_n] + (num_B_regs / 2), acc[mma_id_m][mma_id_n] + (num_acc_regs / 2));
        }
      }
    }
    __syncthreads();

    A += BLOCK_K;
    B += BLOCK_K;
  }

  const int a0_row = lane_id >> 2;
  const int a0_col = (lane_id % 4) * 2;
  C += a0_row * N + a0_col;

  for (int mma_id_m = 0; mma_id_m < NUM_MMA_M; mma_id_m++)
    for (int mma_id_n = 0; mma_id_n < NUM_MMA_N; mma_id_n++) {
      hip_bfloat16 *C_local = C + (mma_id_m * MMA_M) * N + (mma_id_n * MMA_N);
      float *regs = acc[mma_id_m][mma_id_n];

      reinterpret_cast<nv_bfloat162 *>(C_local)[0]             = __float22bfloat162_rn({regs[0], regs[1]});
      reinterpret_cast<nv_bfloat162 *>(C_local + 8 * N)[0]     = __float22bfloat162_rn({regs[2], regs[3]});
      reinterpret_cast<nv_bfloat162 *>(C_local + 8)[0]         = __float22bfloat162_rn({regs[4], regs[5]});
      reinterpret_cast<nv_bfloat162 *>(C_local + 8 * N + 8)[0] = __float22bfloat162_rn({regs[6], regs[7]});
    }
}

void matmul_v5(const hip_bfloat16 *A, const hip_bfloat16 *B, hip_bfloat16 *C, int M, int N, int K) {
  assert(is_power_of_two(M) && "M must be a power of 2");
  assert(is_power_of_two(N) && "N must be a power of 2");
  assert(is_power_of_two(K) && "K must be a power of 2");

  // 4 warps
  // const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 64;  // same as previous kernels
  const int BLOCK_M = 128, BLOCK_N = 64, BLOCK_K = 64; // this is only faster for this kernel
  const int NUM_WARP_M = 2, NUM_WARP_N = 2;

  auto kernel = matmul_v5_kernel<BLOCK_M, BLOCK_N, BLOCK_K, NUM_WARP_M, NUM_WARP_N>;

  const int TB_SIZE = NUM_WARP_M * NUM_WARP_N * WARP_SIZE;
  const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
  const int shm_size = (BLOCK_M + BLOCK_N) * BLOCK_K * sizeof(hip_bfloat16);

  launch_kernel(kernel, grid_size, TB_SIZE, shm_size, A, B, C, M, N, K);
}

template <int BLOCK_M, int BLOCK_N, int BLOCK_K, int NUM_WARP_M, int NUM_WARP_N, int NUM_STAGES>
__launch_bounds__(NUM_WARP_M * NUM_WARP_N * WARP_SIZE) // maxThreadsPerBlock
__global__
void matmul_v6_kernel(const hip_bfloat16 *A, const hip_bfloat16 *B, hip_bfloat16 *C, int M, int N, int K) {
  constexpr int MMA_M = 16;
  constexpr int MMA_N = 16;
  constexpr int MMA_K = 16;
  static_assert(BLOCK_M % NUM_WARP_M == 0);
  static_assert(BLOCK_N % NUM_WARP_N == 0);
  static_assert(BLOCK_K % MMA_K == 0);
  constexpr int WARP_M = BLOCK_M / NUM_WARP_M;
  constexpr int WARP_N = BLOCK_N / NUM_WARP_N;
  static_assert(WARP_M % MMA_M == 0);
  static_assert(WARP_N % MMA_N == 0);
  constexpr int TB_SIZE = NUM_WARP_M * NUM_WARP_N * WARP_SIZE;
  constexpr int NUM_MMA_M = WARP_M / MMA_M;
  constexpr int NUM_MMA_N = WARP_N / MMA_N;
  constexpr int NUM_MMA_K = BLOCK_K / MMA_K;

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int warp_id = tid / WARP_SIZE;
  const int lane_id = tid % WARP_SIZE;

  // TODO: threadblock swizzling to improve L2 cache hit rate
  const int num_blocks_n = cdiv(N, BLOCK_N);
  const int bid_m = bid / num_blocks_n;
  const int bid_n = bid % num_blocks_n;
  const int offset_m = bid_m * BLOCK_M;
  const int offset_n = bid_n * BLOCK_N;

  const int warp_id_m = warp_id / NUM_WARP_N;
  const int warp_id_n = warp_id % NUM_WARP_N;

  // A is row-major, B is column-major, C is row-major
  A += offset_m * K;
  B += offset_n * K;
  C += (offset_m + warp_id_m * WARP_M) * N + (offset_n + warp_id_n * WARP_N);

  // convert shared memory address to 32-bit from the start
  extern __shared__ hip_bfloat16 shm[];
  const uint32_t shm_u32 = cvta_shared(shm);
  const uint32_t A0_shm = shm_u32;
  const uint32_t B0_shm = A0_shm + BLOCK_M * BLOCK_K * sizeof(hip_bfloat16);

  constexpr int num_acc_regs = MMA_M * MMA_N / WARP_SIZE;
  constexpr int num_A_regs = MMA_M * MMA_K * sizeof(hip_bfloat16) / 4 / WARP_SIZE; // 4
  constexpr int num_B_regs = MMA_N * MMA_K * sizeof(hip_bfloat16) / 4 / WARP_SIZE; // 4
  float acc[NUM_MMA_M][NUM_MMA_N][num_acc_regs] = {};
  uint32_t A_regs[NUM_MMA_K][NUM_MMA_M][num_A_regs];
  uint32_t B_regs[NUM_MMA_K][NUM_MMA_N][num_B_regs];

  // pre-compute address used for ldmatrix
  // also pre-compute swizzling
  const int A_offm = (warp_id_m * WARP_M) + (lane_id % 16);
  const int A_offk = (lane_id / 16) * 8;
  const uint32_t A0_shm_thread = swizzle<BLOCK_K * sizeof(hip_bfloat16)>(A0_shm + (A_offm * BLOCK_K + A_offk) * sizeof(hip_bfloat16));

  const int B_offn = (warp_id_n * WARP_N) + (lane_id % 8) + (lane_id / 16) * 8;
  const int B_offk = ((lane_id % 16) / 8) * 8;
  const uint32_t B0_shm_thread = swizzle<BLOCK_K * sizeof(hip_bfloat16)>(B0_shm + (B_offn * BLOCK_K + B_offk) * sizeof(hip_bfloat16));

  // pre-compute the address for each stage
  uint32_t A_shm_thread[NUM_STAGES];
  uint32_t B_shm_thread[NUM_STAGES];
  for (int stage = 0; stage < NUM_STAGES; stage++) {
    A_shm_thread[stage] = A0_shm_thread + stage * (BLOCK_M + BLOCK_N) * BLOCK_K * sizeof(hip_bfloat16);
    B_shm_thread[stage] = B0_shm_thread + stage * (BLOCK_M + BLOCK_N) * BLOCK_K * sizeof(hip_bfloat16);
  }

  // initiate NUM_STAGES-1 async global->shared
  auto global_to_shared = [&](int stage) {
    const uint32_t A_shared = shm_u32 + stage * (BLOCK_M + BLOCK_N) * BLOCK_K * sizeof(hip_bfloat16); // BLOCK_M, BLOCK_K
    const uint32_t B_shared = A_shared + BLOCK_M * BLOCK_K * sizeof(hip_bfloat16);                    // BLOCK_N, BLOCK_K
    global_to_shared_async<TB_SIZE, BLOCK_M, BLOCK_K>(A, K, A_shared, tid);
    global_to_shared_async<TB_SIZE, BLOCK_N, BLOCK_K>(B, K, B_shared, tid);

    // mark this stage as a commit group
    cp_async_commit_group();

    // NOTE: A and B pointers now track position for global->shared load
    A += BLOCK_K;
    B += BLOCK_K;
  };

  // NOTE: we don't take care when num_k_iters < NUM_STAGES
  for (int stage = 0; stage < NUM_STAGES - 1; stage++)
    global_to_shared(stage);

  // loop invariance: there is always NUM_STAGES - 1 prefetch stages in-flight
  // thanks to pipelining, this loop now only has 1 __syncthreads()
  for (int k_iter = 0; k_iter < K / BLOCK_K; k_iter++) {
    if constexpr (NUM_STAGES > 1) {
      // wait for the 1st commit group to finish i.e. FIFO
      // this consumes 1 prefetch
      cp_async_wait_group<NUM_STAGES - 2>();
      __syncthreads(); // why can't we move this after prefetch?

      // prefetch the next stage. restore loop invariance
      // NOTE: to avoid branching here, we can do K / BLOCK_K - NUM_STAGES + 1 in the mainloop
      // and unroll the last NUM_STAGES-1 iterations.
      // NOTE: the location of prefetch in main loop is important.
      // imagine using 2 stages. if we don't issue prefetch immediately after wait_group above,
      // global->shared is not busy anymore. for 3 stages, maybe issue global->shared later is fine?
      const int prefetch_iter = k_iter + NUM_STAGES - 1;
      if (prefetch_iter < (K / BLOCK_K))
        global_to_shared(prefetch_iter % NUM_STAGES);
      else
        cp_async_commit_group();
    } else {
      // without pipelining
      __syncthreads();
      global_to_shared(0);
      cp_async_wait_all();
      __syncthreads();
    }

    const int stage = k_iter % NUM_STAGES;

    // shared->registers
    for (int mma_id_k = 0; mma_id_k < NUM_MMA_K; mma_id_k++) {
      for (int mma_id_m = 0; mma_id_m < NUM_MMA_M; mma_id_m++) {
        const uint32_t A_addr = A_shm_thread[stage] + mma_id_m * MMA_M * BLOCK_K * sizeof(hip_bfloat16);
        ldmatrix<num_A_regs>(A_regs[mma_id_k][mma_id_m], A_addr ^ (mma_id_k * MMA_K * sizeof(hip_bfloat16)));
      }
      for (int mma_id_n = 0; mma_id_n < NUM_MMA_N; mma_id_n++) {
        const uint32_t B_addr = B_shm_thread[stage] + mma_id_n * MMA_N * BLOCK_K * sizeof(hip_bfloat16);
        ldmatrix<num_B_regs>(B_regs[mma_id_k][mma_id_n], B_addr ^ (mma_id_k * MMA_K * sizeof(hip_bfloat16)));
      }
    }

    // do MMA. NUM_STAGES-1 prefetch stages are still on-going
    for (int mma_id_k = 0; mma_id_k < NUM_MMA_K; mma_id_k++)
      for (int mma_id_m = 0; mma_id_m < NUM_MMA_M; mma_id_m++)
        for (int mma_id_n = 0; mma_id_n < NUM_MMA_N; mma_id_n++) {
          uint32_t *A_reg = A_regs[mma_id_k][mma_id_m];
          uint32_t *B_reg = B_regs[mma_id_k][mma_id_n];
          float *acc_reg = acc[mma_id_m][mma_id_n];
          mma(A_reg, B_reg, acc_reg);
          mma(A_reg, B_reg + (num_B_regs / 2), acc_reg + (num_acc_regs / 2));
        }
  }

  const int a0_row = lane_id >> 2;
  const int a0_col = (lane_id % 4) * 2;
  C += a0_row * N + a0_col;

  for (int mma_id_m = 0; mma_id_m < NUM_MMA_M; mma_id_m++)
    for (int mma_id_n = 0; mma_id_n < NUM_MMA_N; mma_id_n++) {
      hip_bfloat16 *C_local = C + (mma_id_m * MMA_M) * N + (mma_id_n * MMA_N);
      float *regs = acc[mma_id_m][mma_id_n];

      reinterpret_cast<nv_bfloat162 *>(C_local)[0]             = __float22bfloat162_rn({regs[0], regs[1]});
      reinterpret_cast<nv_bfloat162 *>(C_local + 8 * N)[0]     = __float22bfloat162_rn({regs[2], regs[3]});
      reinterpret_cast<nv_bfloat162 *>(C_local + 8)[0]         = __float22bfloat162_rn({regs[4], regs[5]});
      reinterpret_cast<nv_bfloat162 *>(C_local + 8 * N + 8)[0] = __float22bfloat162_rn({regs[6], regs[7]});
    }
}

void matmul_v6(const hip_bfloat16 *A, const hip_bfloat16 *B, hip_bfloat16 *C, int M, int N, int K) {
  assert(is_power_of_two(M) && "M must be a power of 2");
  assert(is_power_of_two(N) && "N must be a power of 2");
  assert(is_power_of_two(K) && "K must be a power of 2");

  // 4 warps
  const int BLOCK_M = 128, BLOCK_N = 64, BLOCK_K = 64;
  const int NUM_WARP_M = 2, NUM_WARP_N = 2;
  const int NUM_STAGES = 2;

  auto kernel = matmul_v6_kernel<BLOCK_M, BLOCK_N, BLOCK_K, NUM_WARP_M, NUM_WARP_N, NUM_STAGES>;

  const int TB_SIZE = NUM_WARP_M * NUM_WARP_N * WARP_SIZE;
  const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
  const int shm_size = (BLOCK_M + BLOCK_N) * BLOCK_K * sizeof(hip_bfloat16) * NUM_STAGES;

  launch_kernel(kernel, grid_size, TB_SIZE, shm_size, A, B, C, M, N, K);
}
