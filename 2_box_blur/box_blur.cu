#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <cmath>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__ void box_blur_kernel_v1(const float *input, int kernel_size, float *output, int width, int height) {
  const int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
  const int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

  if (idx_x >= width || idx_y >= height)
    return;

  int radius = (kernel_size - 1) / 2;
  float scale = 1.0f / (kernel_size * kernel_size);
  float total = 0.0f;

  for (int i = max(0, idx_x - radius); i < min(idx_x + radius + 1, width); i++)
    for (int j = max(0, idx_y - radius); j < min(idx_y + radius + 1, height); j++)
      total += input[j * width + i] * scale;

  output[idx_y * width + idx_x] = total;
}

int cdiv(int a, int b) {
  return (a + b - 1) / b;
}

torch::Tensor box_blur_v1(torch::Tensor input, int kernel_size) {
  CHECK_INPUT(input);
  TORCH_CHECK(kernel_size > 0 && kernel_size % 2, "kernel_size must be positive and odd");
  int height = input.size(1);
  int width = input.size(2);
  torch::Tensor output = torch::empty_like(input);

  dim3 n_threads(16, 16);
  dim3 n_blocks(cdiv(width, 16), cdiv(height, 16));
  box_blur_kernel_v1<<<n_blocks, n_threads>>>(input.data_ptr<float>(), kernel_size, output.data_ptr<float>(), width, height);

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("box_blur_v1", &box_blur_v1, "Box blur v1");
}
