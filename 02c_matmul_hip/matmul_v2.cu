#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_bf16.h>

template <int HEIGHT, int WIDTH, int TB_SIZE, typename T>
__device__
void gmem_to_smem(T *dst, const T *src, int src_stride, int tid) {
  using load_type = float4;
  constexpr int multiplier = sizeof(load_type) / sizeof(T);
  static_assert((HEIGHT * WIDTH) % (TB_SIZE * multiplier) == 0);
  constexpr int num_iters = (HEIGHT * WIDTH) / (TB_SIZE * multiplier);

  for (int i = 0; i < num_iters; i++) {
    const int idx = (i * TB_SIZE + tid) * multiplier;
    const int row = idx / WIDTH;
    const int col = idx % WIDTH;

    const load_type data = reinterpret_cast<const load_type *>(src + row * src_stride + col)[0];

    const int swizzled_col = swizzle<WIDTH>(row, col);
    reinterpret_cast<load_type *>(dst + row * WIDTH + swizzled_col)[0] = data;
  }
}

template<int BLOCK_M, int BLOCK_N, int BLOCK_K, int GROUP_M, int NUM_WARP_M, int NUM_WARP_N>
__launch_bounds__(NUM_WARP_M * NUM_WARP_N * WARP_SIZE)
__global__
void matmul_v2_kernel(
  const __hip_bfloat16 *A_gmem,
  const __hip_bfloat16 *B_gmem,
        __hip_bfloat16 *C_gmem,
  int M, int N, int K
) {
  constexpr int WARP_M = BLOCK_M / NUM_WARP_M;
  constexpr int WARP_N = BLOCK_N / NUM_WARP_N;
  constexpr int TB_SIZE = NUM_WARP_M * NUM_WARP_N * WARP_SIZE;

  const int tid = threadIdx.x;
  const int warp_id = tid / WARP_SIZE;
  const int lane_id = tid % WARP_SIZE;

  const int warp_id_m = warp_id / NUM_WARP_N;
  const int warp_id_n = warp_id % NUM_WARP_N;

  const int bid = blockIdx.x;
  const int grid_m = cdiv(M, BLOCK_M);
  const int grid_n = cdiv(N, BLOCK_N);

  int bid_m, bid_n;
  if constexpr (GROUP_M == 1) {
    bid_m = bid / grid_n;
    bid_n = bid % grid_n;
  } else {
    // threadblock swizzling, from triton
    // improve L2 reuse when M is large.
    const int group_size = GROUP_M * grid_n;
    const int group_id = bid / group_size;
    const int first_bid_m = group_id * GROUP_M;
    const int group_size_m = min(grid_m - first_bid_m, GROUP_M);
    bid_m = first_bid_m + ((bid % group_size) % group_size_m);
    bid_n = (bid % group_size) / group_size_m;
  }

  const int offset_m = bid_m * BLOCK_M;
  const int offset_n = bid_n * BLOCK_N;
  A_gmem += offset_m * K;
  B_gmem += offset_n * K;
  C_gmem += (offset_m + warp_id_m * WARP_M) * N + (offset_n + warp_id_n * WARP_N);

  // shared memory
  extern __shared__ __hip_bfloat16 smem[];
  __hip_bfloat16 *A_smem = smem;
  __hip_bfloat16 *B_smem = A_smem + BLOCK_M * BLOCK_K;

  // register memory
  // do it this way to use mfma intrinsic
  s16x4 A_rmem[WARP_M / MMA_M][BLOCK_K / MMA_K];
  s16x4 B_rmem[WARP_N / MMA_N][BLOCK_K / MMA_K];
  fp32x4 C_rmem[WARP_M / MMA_M][WARP_N / MMA_N] = {};

  const int num_k_iters = cdiv(K, BLOCK_K);
  for (int iter_k = 0; iter_k < num_k_iters; iter_k++) {
    // gmem->smem
    __syncthreads();
    gmem_to_smem<BLOCK_M, BLOCK_K, TB_SIZE>(A_smem, A_gmem, K, tid);
    gmem_to_smem<BLOCK_N, BLOCK_K, TB_SIZE>(B_smem, B_gmem, K, tid);
    A_gmem += BLOCK_K;
    B_gmem += BLOCK_K;

    // smem->rmem
    // TODO: use wider load?
    // NOTE: for some reasons, factoring out swizzle out of the main loop is a bit slower.
    __syncthreads();
    for (int mma_id_m = 0; mma_id_m < WARP_M / MMA_M; mma_id_m++)
      for (int mma_id_k = 0; mma_id_k < BLOCK_K / MMA_K; mma_id_k++) {
        const int row = (warp_id_m * WARP_M) + (mma_id_m * MMA_M) + (lane_id % 16);
        const int col = (mma_id_k * MMA_K) + (lane_id / 16) * 4;
        const int swizzled_col = swizzle<BLOCK_K>(row, col);
        __hip_bfloat16 *addr = A_smem + (row * BLOCK_K + swizzled_col);
        A_rmem[mma_id_m][mma_id_k] = reinterpret_cast<s16x4 *>(addr)[0];
      }
    for (int mma_id_n = 0; mma_id_n < WARP_N / MMA_N; mma_id_n++)
      for (int mma_id_k = 0; mma_id_k < BLOCK_K / MMA_K; mma_id_k++) {
        const int row = (warp_id_n * WARP_N) + (mma_id_n * MMA_N) + (lane_id % 16);
        const int col = (mma_id_k * MMA_K) + (lane_id / 16) * 4;
        const int swizzled_col = swizzle<BLOCK_K>(row, col);
        __hip_bfloat16 *addr = B_smem + (row * BLOCK_K + swizzled_col);
        B_rmem[mma_id_n][mma_id_k] = reinterpret_cast<s16x4 *>(addr)[0];
      }

    // mma
    // https://github.com/ROCm/composable_kernel/blob/rocm-7.0.1/include/ck/utility/amd_xdlops.hpp
    // https://github.com/tile-ai/tilelang/blob/v0.1.6.post1/src/tl_templates/hip/gemm.h
    // TODO: swap A and B like in tilelang for better C layout
    for (int mma_id_m = 0; mma_id_m < WARP_M / MMA_M; mma_id_m++)
      for (int mma_id_n = 0; mma_id_n < WARP_N / MMA_N; mma_id_n++)
        for (int mma_id_k = 0; mma_id_k < BLOCK_K / MMA_K; mma_id_k++)
          C_rmem[mma_id_m][mma_id_n] = __builtin_amdgcn_mfma_f32_16x16x16bf16_1k(A_rmem[mma_id_m][mma_id_k],
                                                                                 B_rmem[mma_id_n][mma_id_k],
                                                                                 C_rmem[mma_id_m][mma_id_n],
                                                                                 0, 0, 0);
  }

  __syncthreads();
  for (int mma_id_m = 0; mma_id_m < WARP_M / MMA_M; mma_id_m++)
    for (int mma_id_n = 0; mma_id_n < WARP_N / MMA_N; mma_id_n++) {
      const int row = mma_id_m * MMA_M + (lane_id / 16) * 4;
      const int col = mma_id_n * MMA_N + (lane_id % 16);

      fp32x4 data = C_rmem[mma_id_m][mma_id_n];
      C_gmem[(row + 0) * N + col] = __float2bfloat16(data[0]);
      C_gmem[(row + 1) * N + col] = __float2bfloat16(data[1]);
      C_gmem[(row + 2) * N + col] = __float2bfloat16(data[2]);
      C_gmem[(row + 3) * N + col] = __float2bfloat16(data[3]);
    }
}

void matmul_v2(
  const __hip_bfloat16 *A,
  const __hip_bfloat16 *B,
        __hip_bfloat16 *C,
  int M, int N, int K,
  hipStream_t stream
) {
  constexpr int BLOCK_M = 128;
  constexpr int BLOCK_N = 128;
  constexpr int BLOCK_K = 64;
  constexpr int GROUP_M = 1;

  constexpr int NUM_WARP_M = 2;
  constexpr int NUM_WARP_N = 2;

  const int grid_m = cdiv(M, BLOCK_M);
  const int grid_n = cdiv(N, BLOCK_N);
  const int grid_size = grid_m * grid_n;

  const int tb_size = NUM_WARP_M * NUM_WARP_N * WARP_SIZE;
  const int smem_size = (BLOCK_M + BLOCK_N) * BLOCK_K * sizeof(__hip_bfloat16);

  matmul_v2_kernel<BLOCK_M, BLOCK_N, BLOCK_K, GROUP_M, NUM_WARP_M, NUM_WARP_N>
    <<<grid_size, tb_size, smem_size, stream>>>(A, B, C, M, N, K);
}
