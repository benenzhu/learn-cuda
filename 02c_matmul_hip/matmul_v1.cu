#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_bf16.h>

template <int HEIGHT, int WIDTH, int TB_SIZE, typename T>
__device__
void gmem_to_smem(T *dst, const T *src, int src_stride, int tid) {
  // TODO: figure out optimal load width for MI300X
  constexpr int multiplier = sizeof(float4) / sizeof(T);
  static_assert((HEIGHT * WIDTH) % (TB_SIZE * multiplier) == 0);
  constexpr int num_iters = (HEIGHT * WIDTH) / (TB_SIZE * multiplier);

  for (int i = 0; i < num_iters; i++) {
    const int idx = (i * TB_SIZE + tid) * multiplier;
    const int row = idx / WIDTH;
    const int col = idx % WIDTH;

    const float4 data = reinterpret_cast<const float4 *>(src + row * src_stride + col)[0];
    reinterpret_cast<float4 *>(dst + row * WIDTH + col)[0] = data;
  }
}

template<int BLOCK_M, int BLOCK_N, int BLOCK_K, int GROUP_M, int NUM_WARP_M, int NUM_WARP_N>
__global__
void matmul_v1_kernel(
  const __hip_bfloat16 *A_gmem,
  const __hip_bfloat16 *B_gmem,
        __hip_bfloat16 *C_gmem,
  int M, int N, int K
) {
  constexpr int WARP_M = BLOCK_M / NUM_WARP_M;
  constexpr int WARP_N = BLOCK_N / NUM_WARP_N;
  constexpr int TB_SIZE = NUM_WARP_M * NUM_WARP_N * WARP_SIZE;

  const int tid = threadIdx.x;
  const int warp_id = tid / WARP_SIZE;
  const int lane_id = tid % WARP_SIZE;

  const int warp_id_m = warp_id / NUM_WARP_N;
  const int warp_id_n = warp_id % NUM_WARP_N;

  const int bid = blockIdx.x;
  const int grid_m = cdiv(M, BLOCK_M);
  const int grid_n = cdiv(N, BLOCK_N);

  int bid_m, bid_n;
  if constexpr (GROUP_M == 1) {
    bid_m = bid / grid_n;
    bid_n = bid % grid_n;
  } else {
    // threadblock swizzling, from triton
    // improve L2 reuse when M is large.
    const int group_size = GROUP_M * grid_n;
    const int group_id = bid / group_size;
    const int first_bid_m = group_id * GROUP_M;
    const int group_size_m = min(grid_m - first_bid_m, GROUP_M);
    bid_m = first_bid_m + ((bid % group_size) % group_size_m);
    bid_n = (bid % group_size) / group_size_m;
  }

  const int offset_m = bid_m * BLOCK_M;
  const int offset_n = bid_n * BLOCK_N;
  A_gmem += offset_m * K;
  B_gmem += offset_n * K;
  C_gmem += (offset_m + warp_id_m * WARP_M) * N + (offset_n + warp_id_n * WARP_N);

  // shared memory
  extern __shared__ __hip_bfloat16 smem[];
  __hip_bfloat16 *A_smem = smem;
  __hip_bfloat16 *B_smem = A_smem + BLOCK_M * BLOCK_K;

  // pre-compute offset for smem->rmem
  __hip_bfloat16 *A_smem_thread, *B_smem_thread;
  {
    const int row = (warp_id_m * WARP_M) + (lane_id % 16);
    const int col = (lane_id / 16) * 4;
    A_smem_thread = A_smem + row * BLOCK_K + col;
  }
  {
    const int row = (warp_id_n * WARP_N) + (lane_id % 16);
    const int col = (lane_id / 16) * 4;
    B_smem_thread = B_smem + row * BLOCK_K + col;
  }

  // register memory
  // do it this way to use mfma intrinsic
  my_short4 A_rmem[WARP_M / MMA_M][BLOCK_K / MMA_K];
  my_short4 B_rmem[WARP_N / MMA_N][BLOCK_K / MMA_K];
  my_float4 C_rmem[WARP_M / MMA_M][WARP_N / MMA_N] = {};

  const int num_k_iters = cdiv(K, BLOCK_K);
  for (int iter_k = 0; iter_k < num_k_iters; iter_k++) {
    // gmem->smem
    __syncthreads();
    gmem_to_smem<BLOCK_M, BLOCK_K, TB_SIZE>(A_smem, A_gmem, K, tid);
    gmem_to_smem<BLOCK_N, BLOCK_K, TB_SIZE>(B_smem, B_gmem, K, tid);
    A_gmem += BLOCK_K;
    B_gmem += BLOCK_K;

    // smem->rmem
    // K_L = 16 / (64 / (16 * 1)) = 16 / 4 = 4
    // A: lane (k / 4) * M + i
    // B: lane (k / 4) * N + j
    // TODO: use wider load?
    __syncthreads();
    for (int mma_id_m = 0; mma_id_m < WARP_M / MMA_M; mma_id_m++)
      for (int mma_id_k = 0; mma_id_k < BLOCK_K / MMA_K; mma_id_k++) {
        const int row = mma_id_m * MMA_M;
        const int col = mma_id_k * MMA_K;
        __hip_bfloat16 *addr = A_smem_thread + row * BLOCK_K + col;
        A_rmem[mma_id_m][mma_id_k] = reinterpret_cast<my_short4 *>(addr)[0];
      }
    for (int mma_id_n = 0; mma_id_n < WARP_N / MMA_N; mma_id_n++)
      for (int mma_id_k = 0; mma_id_k < BLOCK_K / MMA_K; mma_id_k++) {
        const int row = mma_id_n * MMA_N;
        const int col = mma_id_k * MMA_K;
        __hip_bfloat16 *addr = B_smem_thread + row * BLOCK_K + col;
        B_rmem[mma_id_n][mma_id_k] = reinterpret_cast<my_short4 *>(addr)[0];
      }

    // mma
    // https://github.com/ROCm/composable_kernel/blob/rocm-7.0.1/include/ck/utility/amd_xdlops.hpp
    // https://github.com/tile-ai/tilelang/blob/v0.1.6.post1/src/tl_templates/hip/gemm.h
    // TODO: swap A and B like in tilelang for better C layout
    for (int mma_id_m = 0; mma_id_m < WARP_M / MMA_M; mma_id_m++)
      for (int mma_id_n = 0; mma_id_n < WARP_N / MMA_N; mma_id_n++)
        for (int mma_id_k = 0; mma_id_k < BLOCK_K / MMA_K; mma_id_k++)
          C_rmem[mma_id_m][mma_id_n] = __builtin_amdgcn_mfma_f32_16x16x16bf16_1k(A_rmem[mma_id_m][mma_id_k],
                                                                                 B_rmem[mma_id_n][mma_id_k],
                                                                                 C_rmem[mma_id_m][mma_id_n],
                                                                                 0, 0, 0);
  }

  // H = 4
  // B_I = ceil(64 / (N * M / H)) = ceil(64 / (16 * 16 / 4)) = 1
  // M_I = (64 / B_I) / N = 4
  // G = M / (H * M_I) = 1
  // item: (i % 4)
  // lane: (i / 4) * N + j
  __syncthreads();
  for (int mma_id_m = 0; mma_id_m < WARP_M / MMA_M; mma_id_m++)
    for (int mma_id_n = 0; mma_id_n < WARP_N / MMA_N; mma_id_n++) {
      const int row = mma_id_m * MMA_M + (lane_id / 16) * 4;
      const int col = mma_id_n * MMA_N + (lane_id % 16);

      my_float4 data = C_rmem[mma_id_m][mma_id_n];
      C_gmem[(row + 0) * N + col] = __float2bfloat16(data[0]);
      C_gmem[(row + 1) * N + col] = __float2bfloat16(data[1]);
      C_gmem[(row + 2) * N + col] = __float2bfloat16(data[2]);
      C_gmem[(row + 3) * N + col] = __float2bfloat16(data[3]);
    }
}

void matmul_v1(
  const __hip_bfloat16 *A,
  const __hip_bfloat16 *B,
        __hip_bfloat16 *C,
  int M, int N, int K,
  hipStream_t stream
) {
  constexpr int BLOCK_M = 128;
  constexpr int BLOCK_N = 128;
  constexpr int BLOCK_K = 64;
  constexpr int GROUP_M = 1;

  constexpr int NUM_WARP_M = 2;
  constexpr int NUM_WARP_N = 2;

  const int grid_m = cdiv(M, BLOCK_M);
  const int grid_n = cdiv(N, BLOCK_N);
  const int grid_size = grid_m * grid_n;

  const int tb_size = NUM_WARP_M * NUM_WARP_N * WARP_SIZE;
  const int smem_size = (BLOCK_M + BLOCK_N) * BLOCK_K * sizeof(__hip_bfloat16);

  matmul_v1_kernel<BLOCK_M, BLOCK_N, BLOCK_K, GROUP_M, NUM_WARP_M, NUM_WARP_N>
    <<<grid_size, tb_size, smem_size, stream>>>(A, B, C, M, N, K);
}
