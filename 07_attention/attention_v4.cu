#include "hip/hip_runtime.h"
#include "common.h"

#include <hip/hip_bf16.h>
#include <cstdint>
#include <float.h>
#include <iostream>

template<int BLOCK_Q, int BLOCK_KV, int DIM, int NUM_WARPS>
__launch_bounds__(NUM_WARPS * WARP_SIZE)
__global__
void attention_v4_kernel(
  const hip_bfloat16 *Q,  // [bs, len_q, DIM]
  const hip_bfloat16 *K,  // [bs, len_kv, DIM]
  const hip_bfloat16 *V,  // [bs, len_kv, DIM]
  hip_bfloat16 *O,        // [bs, len_q, DIM]
  int bs,
  int len_q,
  int len_kv) {

  constexpr int TB_SIZE = NUM_WARPS * WARP_SIZE;

  const int bid = blockIdx.x;
  const int tid = threadIdx.x;
  const int warp_id = tid / WARP_SIZE;
  const int lane_id = tid % WARP_SIZE;

  // each threadblock handles 1 BLOCK_Q
  const int num_q_blocks = cdiv(len_q, BLOCK_Q);
  const int bs_id = bid / num_q_blocks;
  const int q_block_id = bid % num_q_blocks;

  Q += (bs_id * num_q_blocks + q_block_id) * BLOCK_Q * DIM;
  K += bs_id * len_kv * DIM;
  V += bs_id * len_kv * DIM;
  O += (bs_id * num_q_blocks + q_block_id) * BLOCK_Q * DIM;

  // we overlap Q_smem with (K_smem + V_smem), since we only need to load Q_smem once
  extern __shared__ hip_bfloat16 smem[];
  const uint32_t Q_smem = __cvta_generic_to_shared(smem);
  const uint32_t K_smem = Q_smem;
  const uint32_t V_smem = K_smem + BLOCK_KV * DIM * sizeof(hip_bfloat16);

  // FA2: shard BLOCK_Q among all warps
  // replicate K and V on all warps
  constexpr int WARP_Q = BLOCK_Q / NUM_WARPS;

  // mma.m16n8k16
  constexpr int MMA_M = 16;
  constexpr int MMA_N = 8;
  constexpr int MMA_K = 16;

  // set up registers
  uint32_t Q_rmem[WARP_Q / MMA_M][DIM / MMA_K][4];
  uint32_t K_rmem[BLOCK_KV / MMA_N][DIM / MMA_K][2];

  // let compiler decide register reuse?
  uint32_t P_rmem[WARP_Q / MMA_M][BLOCK_KV / MMA_K][4];
  uint32_t V_rmem[BLOCK_KV / MMA_K][DIM / MMA_N][2];

  // rescale O_rmem once we obtain new rowmax, then accumulate to O_rmem for P @ V
  float O_rmem[WARP_Q / MMA_M][DIM / MMA_N][4] = {};

  // pre-compute address and swizzling for ldmatrix
  uint32_t Q_smem_thread, K_smem_thread, V_smem_thread;
  {
    // A tile
    const int row_off = warp_id * WARP_Q + (lane_id % 16);
    const int col_off = lane_id / 16 * 8;
    Q_smem_thread = swizzle<DIM * sizeof(hip_bfloat16)>(Q_smem + (row_off * DIM + col_off) * sizeof(hip_bfloat16));
  }
  {
    // B tile
    const int row_off = lane_id % 8;
    const int col_off = lane_id / 8 * 8;
    K_smem_thread = swizzle<DIM * sizeof(hip_bfloat16)>(K_smem + (row_off * DIM + col_off) * sizeof(hip_bfloat16));
  }
  {
    // B tile trans
    const int row_off = lane_id % 16;
    const int col_off = lane_id / 16 * 8;
    V_smem_thread = swizzle<DIM * sizeof(hip_bfloat16)>(V_smem + (row_off * DIM + col_off) * sizeof(hip_bfloat16));
  }

  const float softmax_scale = rsqrtf(static_cast<float>(DIM));

  float rowmax[WARP_Q / MMA_M][2];
  float rowsumexp[WARP_Q / MMA_M][2] = {};

  for (int mma_id_q = 0; mma_id_q < WARP_Q / MMA_M; mma_id_q++) {
    rowmax[mma_id_q][0] = -FLT_MAX;
    rowmax[mma_id_q][1] = -FLT_MAX;
  }

  // load Q [BLOCK_Q, DIM]
  global_to_shared_swizzle<BLOCK_Q, DIM, TB_SIZE>(Q_smem, Q, DIM, tid);
  asm volatile("cp.async.commit_group;");
  asm volatile("cp.async.wait_all;");
  __syncthreads();

  // shared -> registers
  for (int mma_id_q = 0; mma_id_q < WARP_Q / MMA_M; mma_id_q++)
    for (int mma_id_d = 0; mma_id_d < DIM / MMA_K; mma_id_d++) {
      uint32_t addr = Q_smem_thread;
      addr += mma_id_q * MMA_M * DIM * sizeof(hip_bfloat16);  // row
      addr ^= mma_id_d * MMA_K * sizeof(hip_bfloat16);  // col
      ldmatrix_x4(Q_rmem[mma_id_q][mma_id_d], addr);
    }
  // we need a syncthreads() here so that we don't load K global->shared
  // before finishing loading Q shared->reg
  __syncthreads();

  const int num_kv_iter = cdiv(len_kv, BLOCK_KV);

  auto load_K = [&](int kv_id) {
    if (kv_id < num_kv_iter) {
      const uint32_t dst = K_smem + (kv_id % 2) * (2 * BLOCK_KV * DIM * sizeof(hip_bfloat16));
      global_to_shared_swizzle<BLOCK_KV, DIM, TB_SIZE>(dst, K, DIM, tid);
      K += BLOCK_KV * DIM;
    }
    asm volatile("cp.async.commit_group;");
  };
  auto load_V = [&](int kv_id) {
    if (kv_id < num_kv_iter) {
      const uint32_t dst = V_smem + (kv_id % 2) * (2 * BLOCK_KV * DIM * sizeof(hip_bfloat16));
      global_to_shared_swizzle<BLOCK_KV, DIM, TB_SIZE>(dst, V, DIM, tid);
      V += BLOCK_KV * DIM;
    }
    asm volatile("cp.async.commit_group;");
  };

  // prefetch K and V
  load_K(0);
  load_V(0);

  for (int kv_id = 0; kv_id < num_kv_iter; kv_id++) {
    float S_rmem[WARP_Q / MMA_M][BLOCK_KV / MMA_N][4] = {};

    // prefetch K
    load_K(kv_id + 1);
    asm volatile("cp.async.wait_group 2;");
    __syncthreads();

    // shared -> registers
    for (int mma_id_kv = 0; mma_id_kv < BLOCK_KV / MMA_N; mma_id_kv++)
      for (int mma_id_d = 0; mma_id_d < DIM / MMA_K; mma_id_d += 2) {
        uint32_t addr = K_smem_thread + (kv_id % 2) * (2 * BLOCK_KV * DIM * sizeof(hip_bfloat16));
        addr += mma_id_kv * MMA_N * DIM * sizeof(hip_bfloat16);  // row
        addr ^= mma_id_d * MMA_K * sizeof(hip_bfloat16);  // col
        ldmatrix_x4(K_rmem[mma_id_kv][mma_id_d], addr);
      }

    // MMA S = Q @ K.T [BLOCK_Q, BLOCK_KV]
    for (int mma_id_q = 0; mma_id_q < WARP_Q / MMA_M; mma_id_q++)
      for (int mma_id_kv = 0; mma_id_kv < BLOCK_KV / MMA_N; mma_id_kv++)
        for (int mma_id_d = 0; mma_id_d < DIM / MMA_K; mma_id_d++)
          mma_m16n8k16(Q_rmem[mma_id_q][mma_id_d],
                       K_rmem[mma_id_kv][mma_id_d],
                       S_rmem[mma_id_q][mma_id_kv]);

    // prefetch V
    load_V(kv_id + 1);

    for (int mma_id_q = 0; mma_id_q < WARP_Q / MMA_M; mma_id_q++) {
      // apply softmax scale
      for (int mma_id_kv = 0; mma_id_kv < BLOCK_KV / MMA_N; mma_id_kv++)
        for (int reg_id = 0; reg_id < 4; reg_id++)
          S_rmem[mma_id_q][mma_id_kv][reg_id] *= softmax_scale;

      // rowmax
      float this_rowmax[2];
      for (int mma_id_kv = 0; mma_id_kv < BLOCK_KV / MMA_N; mma_id_kv++) {
        float *regs = S_rmem[mma_id_q][mma_id_kv];
        if (mma_id_kv == 0) {
          this_rowmax[0] = max(regs[0], regs[1]);  // c0 and c1
          this_rowmax[1] = max(regs[2], regs[3]);  // c2 and c3
        } else {
          this_rowmax[0] = max(this_rowmax[0], max(regs[0], regs[1]));  // c0 and c1
          this_rowmax[1] = max(this_rowmax[1], max(regs[2], regs[3]));  // c2 and c3
        }
      }

      // butterfly reduction within 4 threads
      this_rowmax[0] = max(this_rowmax[0], __shfl_xor_sync(0xFFFF'FFFF, this_rowmax[0], 1));
      this_rowmax[0] = max(this_rowmax[0], __shfl_xor_sync(0xFFFF'FFFF, this_rowmax[0], 2));
      this_rowmax[1] = max(this_rowmax[1], __shfl_xor_sync(0xFFFF'FFFF, this_rowmax[1], 1));
      this_rowmax[1] = max(this_rowmax[1], __shfl_xor_sync(0xFFFF'FFFF, this_rowmax[1], 2));

      // new rowmax
      this_rowmax[0] = max(this_rowmax[0], rowmax[mma_id_q][0]);
      this_rowmax[1] = max(this_rowmax[1], rowmax[mma_id_q][1]);

      // rescale for previous O
      float rescale[2];
      rescale[0] = __expf(rowmax[mma_id_q][0] - this_rowmax[0]);
      rescale[1] = __expf(rowmax[mma_id_q][1] - this_rowmax[1]);
      for (int mma_id_d = 0; mma_id_d < DIM / MMA_N; mma_id_d++) {
        O_rmem[mma_id_q][mma_id_d][0] *= rescale[0];
        O_rmem[mma_id_q][mma_id_d][1] *= rescale[0];
        O_rmem[mma_id_q][mma_id_d][2] *= rescale[1];
        O_rmem[mma_id_q][mma_id_d][3] *= rescale[1];
      }

      // save new rowmax
      rowmax[mma_id_q][0] = this_rowmax[0];
      rowmax[mma_id_q][1] = this_rowmax[1];

      // rowsumexp
      float this_rowsumexp[2];
      for (int mma_id_kv = 0; mma_id_kv < BLOCK_KV / MMA_N; mma_id_kv++) {
        float *regs = S_rmem[mma_id_q][mma_id_kv];
        regs[0] = __expf(regs[0] - rowmax[mma_id_q][0]);  // c0
        regs[1] = __expf(regs[1] - rowmax[mma_id_q][0]);  // c1
        regs[2] = __expf(regs[2] - rowmax[mma_id_q][1]);  // c2
        regs[3] = __expf(regs[3] - rowmax[mma_id_q][1]);  // c3

        if (mma_id_kv == 0) {
          this_rowsumexp[0] = regs[0] + regs[1];
          this_rowsumexp[1] = regs[2] + regs[3];
        } else {
          this_rowsumexp[0] += regs[0] + regs[1];
          this_rowsumexp[1] += regs[2] + regs[3];
        }

        // pack to P registers for next MMA
        // we need to change from m16n8 to m16k16
        nv_bfloat162 *this_P_rmem = reinterpret_cast<nv_bfloat162 *>(P_rmem[mma_id_q][mma_id_kv / 2]);
        this_P_rmem[(mma_id_kv % 2) * 2]     = __float22bfloat162_rn({regs[0], regs[1]});
        this_P_rmem[(mma_id_kv % 2) * 2 + 1] = __float22bfloat162_rn({regs[2], regs[3]});
      }

      // butterfly reduction within 4 threads
      this_rowsumexp[0] += __shfl_xor_sync(0xFFFF'FFFF, this_rowsumexp[0], 1);
      this_rowsumexp[0] += __shfl_xor_sync(0xFFFF'FFFF, this_rowsumexp[0], 2);
      this_rowsumexp[1] += __shfl_xor_sync(0xFFFF'FFFF, this_rowsumexp[1], 1);
      this_rowsumexp[1] += __shfl_xor_sync(0xFFFF'FFFF, this_rowsumexp[1], 2);

      // accumulate to total rowsumexp
      rowsumexp[mma_id_q][0] = rowsumexp[mma_id_q][0] * rescale[0] + this_rowsumexp[0];
      rowsumexp[mma_id_q][1] = rowsumexp[mma_id_q][1] * rescale[1] + this_rowsumexp[1];
    }

    // wait V load to finish
    asm volatile("cp.async.wait_group 2;");
    __syncthreads();

    // shared -> registers
    for (int mma_id_kv = 0; mma_id_kv < BLOCK_KV / MMA_K; mma_id_kv++)
      for (int mma_id_d = 0; mma_id_d < DIM / MMA_N; mma_id_d += 2) {
        uint32_t addr = V_smem_thread + (kv_id % 2) * (2 * BLOCK_KV * DIM * sizeof(hip_bfloat16));
        addr += mma_id_kv * MMA_K * DIM * sizeof(hip_bfloat16);  // row
        addr ^= mma_id_d * MMA_N * sizeof(hip_bfloat16);  // col
        ldmatrix_x4_trans(V_rmem[mma_id_kv][mma_id_d], addr);
      }

    // MMA P = S @ V [BLOCK_Q, DIM]
    for (int mma_id_q = 0; mma_id_q < WARP_Q / MMA_M; mma_id_q++)
      for (int mma_id_d = 0; mma_id_d < DIM / MMA_N; mma_id_d++)
        for (int mma_id_kv = 0; mma_id_kv < BLOCK_KV / MMA_K; mma_id_kv++)
          mma_m16n8k16(P_rmem[mma_id_q][mma_id_kv],
                       V_rmem[mma_id_kv][mma_id_d],
                       O_rmem[mma_id_q][mma_id_d]);
  }

  // write to O
  for (int mma_id_q = 0; mma_id_q < WARP_Q / MMA_M; mma_id_q++)
    for (int mma_id_d = 0; mma_id_d < DIM / MMA_N; mma_id_d++) {
      const int row = warp_id * WARP_Q + mma_id_q * MMA_M + (lane_id / 4);
      const int col = mma_id_d * MMA_N + (lane_id % 4) * 2;

      // divide by softmax denominator
      float *regs = O_rmem[mma_id_q][mma_id_d];
      regs[0] /= rowsumexp[mma_id_q][0];
      regs[1] /= rowsumexp[mma_id_q][0];
      regs[2] /= rowsumexp[mma_id_q][1];
      regs[3] /= rowsumexp[mma_id_q][1];

      reinterpret_cast<nv_bfloat162 *>(O + (row + 0) * DIM + col)[0] = __float22bfloat162_rn({regs[0], regs[1]});
      reinterpret_cast<nv_bfloat162 *>(O + (row + 8) * DIM + col)[0] = __float22bfloat162_rn({regs[2], regs[3]});
    }
}

void attention_v4(
  const hip_bfloat16 *Q,  // [bs, len_q, DIM]
  const hip_bfloat16 *K,  // [bs, len_kv, DIM]
  const hip_bfloat16 *V,  // [bs, len_kv, DIM]
  hip_bfloat16 *O,        // [bs, len_q, DIM]
  int bs,
  int len_q,
  int len_kv,
  int dim) {

  if (dim != 128) {
    std::cerr << "Unsupported dim=" << dim << std::endl;
    exit(1);
  }

  const int BLOCK_Q = 64;
  const int BLOCK_KV = 32;
  const int DIM = 128;
  const int NUM_WARPS = 4;

  const int num_blocks = bs * cdiv(len_q, BLOCK_Q);
  const int TB_SIZE = NUM_WARPS * WARP_SIZE;
  const int smem_size = max(BLOCK_Q, BLOCK_KV * 2 * 2) * DIM * sizeof(hip_bfloat16);

  auto kernel = attention_v4_kernel<BLOCK_Q, BLOCK_KV, DIM, NUM_WARPS>;
  launch_kernel(kernel, num_blocks, TB_SIZE, smem_size, Q, K, V, O, bs, len_q, len_kv);
}
