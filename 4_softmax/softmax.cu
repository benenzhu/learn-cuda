
#include <hip/hip_runtime.h>
__host__ __device__ int cdiv(int a, int b) { return (a + b - 1) / b; }
__device__ float add(float a, float b) { return a + b; }

constexpr int WARP_SIZE = 32;

template <float f(float, float)>
__device__ float thread_reduce(float val, const float *input, int TILE_SIZE, int BLOCK_SIZE, int tid, int max_idx) {
  for (int idx = tid; idx < TILE_SIZE; idx += BLOCK_SIZE)
    if (idx < max_idx)
      val = f(val, input[idx]);
  return val;
}

template <float f(float, float)>
__device__ float block_reduce(float val, int BLOCK_SIZE, int tid, float *reduce_space) {
  reduce_space[tid] = val;
  for (int stride = BLOCK_SIZE / 2; stride >= WARP_SIZE; stride /= 2) {
    __syncthreads();
    if (tid < stride) {
      val = f(val, reduce_space[tid + stride]);
      reduce_space[tid] = val;
    }
  }
  return val;
}

template <float f(float, float)>
__device__ float warp_reduce(float val, int tid) {
  if (tid < WARP_SIZE)
    for (int stride = WARP_SIZE / 2; stride > 0; stride /= 2)
      val = f(val, __shfl_down_sync(0xffffffff, val, stride));
  return val;
}

__device__ float block_broadcast(float val, int tid, float *shmem) {
  if (tid == 0)
    shmem[0] = val;
  __syncthreads();
  return shmem[0];
}

// https://stackoverflow.com/a/72461459
// when val > 0, use atomicMax signed int. in sint representation:
//   - -ve float < +ve float.
//   - less +ve float < more +ve float.
// when val < 0, use atomicMin unsigned int. in uint representation:
//   - +ve float < -ve float.
//   - less -ve float < more -ve float.
// we use !signbit(value) instead of (value > 0) because there is -0 in float.
__device__ float atomicMax(float *address, float val) {
  return !signbit(val) ? 
    __int_as_float(atomicMax(reinterpret_cast<int *>(address), __float_as_int(val))) :
    __uint_as_float(atomicMin(reinterpret_cast<unsigned int*>(address), __float_as_uint(val)));
}

__global__ void softmax_v1_kernel_pass1(const float *input, float *max_space, int M, int N, int TILE_SIZE) {
  const int tid = threadIdx.x;
  const int BLOCK_SIZE = blockDim.x;
  const int tile_id = blockIdx.x;
  const int row = blockIdx.y;

  input += row * N + tile_id * TILE_SIZE;
  max_space += row;

  extern __shared__ float reduce_space[];

  float max_val = -INFINITY;
  max_val = thread_reduce<max>(max_val, input, TILE_SIZE, BLOCK_SIZE, tid, N);
  max_val = block_reduce<max>(max_val, BLOCK_SIZE, tid, reduce_space);
  max_val = warp_reduce<max>(max_val, tid);

  // TODO: atomicMax is not implemented for float
  if (tid == 0)
    atomicMax(max_space, max_val);
}

__global__ void softmax_v1_kernel_pass2(const float *input, float *output, float *max_space, float *normalizer_space, int M, int N, int TILE_SIZE) {
  const int tid = threadIdx.x;
  const int BLOCK_SIZE = blockDim.x;
  const int tile_id = blockIdx.x;
  const int row = blockIdx.y;

  input += row * N + tile_id * TILE_SIZE;
  output += row * N + tile_id * TILE_SIZE;
  max_space += row;
  normalizer_space += row;

  extern __shared__ float reduce_space[];

  float sum = 0.0f;
  float subtract = max_space[0];
  for (int idx = tid; idx < TILE_SIZE; idx += BLOCK_SIZE)
    if (idx < N) {
      float val = exp(input[idx] - subtract);
      output[idx] = val;
      sum += val;
    }
  sum = block_reduce<add>(sum, BLOCK_SIZE, tid, reduce_space);
  sum = warp_reduce<add>(sum, tid);
  if (tid == 0)
    atomicAdd(normalizer_space, sum);
}

__global__ void softmax_v1_kernel_pass3(float *output, float *normalizer_space, int M, int N, int TILE_SIZE) {
  const int tid = threadIdx.x;
  const int BLOCK_SIZE = blockDim.x;
  const int tile_id = blockIdx.x;
  const int row = blockIdx.y;

  output += row * N + tile_id * TILE_SIZE;
  normalizer_space += row;

  float scale = 1.0f / normalizer_space[0];
  for (int idx = tid; idx < TILE_SIZE; idx += BLOCK_SIZE)
    if (idx < N)
      output[idx] *= scale;
}

void softmax_v1(const float *input, float *output, float *workspace, int M, int N) {
  // need extra 2M space to store max per row and sum per row
  const int BLOCK_SIZE = 256;
  const int TILE_SIZE = BLOCK_SIZE * 4;
  const dim3 grid_size(cdiv(N, TILE_SIZE), M);
  const int reduce_space_size = sizeof(float) * BLOCK_SIZE;

  float *max_space = workspace;
  float *normalizer_space = workspace + M;

  // pass 1: max per row
  softmax_v1_kernel_pass1<<<grid_size, BLOCK_SIZE, reduce_space_size>>>(input, max_space, M, N, TILE_SIZE);

  // pass 2: exp(x - max) and sum
  softmax_v1_kernel_pass2<<<grid_size, BLOCK_SIZE, reduce_space_size>>>(input, output, max_space, normalizer_space, M, N, TILE_SIZE);

  // pass 3: normalize
  softmax_v1_kernel_pass3<<<grid_size, BLOCK_SIZE>>>(output, normalizer_space, M, N, TILE_SIZE);
}

template <bool STORE_INTERMEDIATE>
__global__ void softmax_v2_kernel(const float *input, float *output, int M, int N) {
  const int tid = threadIdx.x;
  const int BLOCK_SIZE = blockDim.x;
  const int row = blockIdx.y;

  input += row * N;
  output += row * N;

  extern __shared__ float shmem_reduce[];

  // pass 1: find max
  float max_val = -INFINITY;
  max_val = thread_reduce<max>(max_val, input, N, BLOCK_SIZE, tid, N);
  max_val = block_reduce<max>(max_val, BLOCK_SIZE, tid, shmem_reduce);
  max_val = warp_reduce<max>(max_val, tid);
  max_val = block_broadcast(max_val, tid, shmem_reduce);

  // pass 2: subtract max and apply exponential + find sum
  float sum = 0.0f;
  for (int col = tid; col < N; col += BLOCK_SIZE) {
    float val = exp(input[col] - max_val);
    sum += val;
    if (STORE_INTERMEDIATE)
      output[col] = val;
  }
  sum = block_reduce<add>(sum, BLOCK_SIZE, tid, shmem_reduce);
  sum = warp_reduce<add>(sum, tid);
  sum = block_broadcast(sum, tid, shmem_reduce);

  // pass 3: normalize
  // NOTE: if N is small, we can cache exp(input[col] - max_val) in shared memory
  float normalizer = 1.0f / sum;
  for (int col = tid; col < N; col += BLOCK_SIZE)
    output[col] = (STORE_INTERMEDIATE ? output[col] : exp(input[col] - max_val)) * normalizer;
}

void softmax_v2a(const float *input, float *output, int M, int N) {
  const int BLOCK_SIZE = 1024;
  const dim3 grid_size(1, M);
  const int shmem_size = sizeof(float) * BLOCK_SIZE;
  softmax_v2_kernel<false><<<grid_size, BLOCK_SIZE, shmem_size>>>(input, output, M, N);
}

void softmax_v2b(const float *input, float *output, int M, int N) {
  const int BLOCK_SIZE = 1024;
  const dim3 grid_size(1, M);
  const int shmem_size = sizeof(float) * BLOCK_SIZE;
  softmax_v2_kernel<true><<<grid_size, BLOCK_SIZE, shmem_size>>>(input, output, M, N);
}

__global__ void softmax_v3_kernel_pass1(const float *input, float *workspace, int M, int N, int TILE_SIZE) {
  const int tid = threadIdx.x;
  const int BLOCK_SIZE = blockDim.x;
  const int tile_id = blockIdx.x;
  const int row = blockIdx.y;

  input += row * N + tile_id * TILE_SIZE;
  float *max_space = workspace + row;
  float *normalizer_space = workspace + M + row;

  extern __shared__ float max_shared[];
  float *normalizer_shared = max_shared + BLOCK_SIZE;

  // algorithm 3 in https://arxiv.org/pdf/1805.02867
  float max_val = -INFINITY;
  float normalizer = 0.0f;
  for (int idx = tid; idx < TILE_SIZE; idx += BLOCK_SIZE) {
    float val = input[idx];
    float old_max_val = max_val;
    max_val = max(max_val, val);
    normalizer = normalizer * exp(old_max_val - max_val) + exp(val - max_val);
  }

  // equation 4, section 3.1 in https://arxiv.org/pdf/1805.02867
  max_shared[tid] = max_val;
  normalizer_shared[tid] = normalizer;
  for (int stride = BLOCK_SIZE / 2; stride < WARP_SIZE; stride /= 2) {
    __syncthreads();
    if (tid < stride) {
      float other_max = max_shared[tid + stride];
      float other_normalizer = normalizer_shared[tid + stride];

      float new_max = max(max_val, other_max);
      normalizer = normalizer * exp(max_val - new_max) + other_normalizer * exp(other_max - new_max);
      max_val = new_max;
      max_shared[tid] = new_max;
      normalizer_shared[tid] = normalizer;
    }
  }

  if (tid < WARP_SIZE)
    for (int stride = WARP_SIZE / 2; stride > 0; stride /= 2) {
      float other_max = __shfl_down_sync(0xffffffff, max_val, stride);
      float other_normalizer = __shfl_down_sync(0xffffffff, normalizer, stride);

      float new_max = max(max_val, other_max);
      normalizer = normalizer * exp(max_val - new_max) + other_normalizer * exp(other_max - new_max);
      max_val = new_max;
    }

  if (tid == 0) {
    float other_max = atomicMax(max_space, max_val);
    float new_max = max(max_val, other_max);

    // using atomicCAS is slow
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
    float other_normalizer = normalizer_space[0];
    float assumed;
    do {
      assumed = other_normalizer;
      float new_normalizer = normalizer * exp(max_val - new_max) + other_normalizer * exp(other_max - new_max);
      other_normalizer = atomicCAS(reinterpret_cast<int *>(normalizer_space), __float_as_int(other_normalizer), __float_as_int(new_normalizer));
    } while (assumed != other_normalizer);
  }
}

__global__ void softmax_v3_kernel_pass2(const float *input, float *output, const float *workspace, int M, int N, int TILE_SIZE) {
  const int tid = threadIdx.x;
  const int BLOCK_SIZE = blockDim.x;
  const int tile_id = blockIdx.x;
  const int row = blockIdx.y;

  input += row * N + tile_id * TILE_SIZE;
  output += row * N + tile_id * TILE_SIZE;
  float row_max  = workspace[row];
  float scale = 1.0f / workspace[M + row];

  for (int idx = tid; idx < TILE_SIZE; idx += BLOCK_SIZE)
    if (idx < N)
      output[idx] = exp(input[idx] - row_max) * scale;
}

// online softmax
void softmax_v3(const float *input, float *output, float *workspace, int M, int N) {
  const int BLOCK_SIZE = 256;
  const int TILE_SIZE = BLOCK_SIZE * 4;
  const dim3 grid_size(cdiv(N, BLOCK_SIZE), M);
  const int shmem_size = sizeof(float) * BLOCK_SIZE * 2;

  // pass 1: find max and normalizer at the same time
  softmax_v3_kernel_pass1<<<grid_size, BLOCK_SIZE, shmem_size>>>(input, workspace, M, N, TILE_SIZE);

  // pass 2: calculate output
  softmax_v3_kernel_pass2<<<grid_size, BLOCK_SIZE>>>(input, output, workspace, M, N, TILE_SIZE);
}
