#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <cmath>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define cdiv(a, b) ((a) + (b) - 1) / (b)

// Kahan sum to reduce errors
__global__ void sum_kernel_v1(const float *input, float *output, int m, int n) {
  const int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (row_idx >= m)
    return;

  float sum = 0.0f;
  float error = 0.0f;

  for (int i = 0; i < n; i++) {
    float item = input[row_idx * n + i] - error;
    float new_sum = sum + item;
    error = new_sum - sum - item;
    sum = new_sum;
  }
  
  output[row_idx] = sum;
}

torch::Tensor sum_v1(torch::Tensor input) {
  CHECK_INPUT(input);
  int m = input.size(0);
  int n = input.size(1);
  torch::Tensor output = torch::empty({m}, input.options());

  int n_threads = 256;
  int n_blocks = cdiv(m, n_threads);
  sum_kernel_v1<<<n_blocks, n_threads>>>(input.data_ptr<float>(), output.data_ptr<float>(), m, n);

  return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("sum_v1", &sum_v1, "Sum v1");
}
