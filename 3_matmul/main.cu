#include <hip/hip_runtime.h>
#include <iostream>

void matmul_v1(const float *A, const float *B, float *C, int M, int N, int K);
void matmul_v2(const float *A, const float *B, float *C, int M, int N, int K);

int main() {
  // Size of the input data
  const int N = 4096;

  // Allocate memory for input and output on host
  float *A = new float[N * N];
  float *B = new float[N * N];
  float *C = new float[N * N];

  // Initialize input data on host
  for (int i = 0; i < N * N; i++) {
    A[i] = 1.0f; // Example: Initialize all elements to 1
    B[i] = 1.0f; // Example: Initialize all elements to 1
  }

  // Allocate memory for input and output on device
  float *d_A;
  float *d_B;
  float *d_C;

  hipMalloc(&d_A, N * N * sizeof(float));
  hipMalloc(&d_B, N * N * sizeof(float));
  hipMalloc(&d_C, N * N * sizeof(float));

  // Copy data from host to devic
  hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

  // Launch the kernel
  // matmul_v1(d_A, d_B, d_C, N, N, N);
  matmul_v2(d_A, d_B, d_C, N, N, N);

  // Copy result back to host
  hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

  // Check results
  for (int col = 0; col < N; col++)
    for (int row = 0; row < N; row++)
      if (C[row * N + col] != N)
        std::cout << "Wrong result at (" << row << ", " << col << ")" << std::endl;

  // Cleanup
  delete[] A;
  delete[] B;
  delete[] C;
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return 0;
}
